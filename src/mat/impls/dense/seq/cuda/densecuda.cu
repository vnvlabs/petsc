#include "hip/hip_runtime.h"
/*
     Defines the matrix operations for sequential dense with CUDA
*/
#include <petscpkg_version.h>
#define PETSC_SKIP_IMMINTRIN_H_CUDAWORKAROUND 1
#include <../src/mat/impls/dense/seq/dense.h> /*I "petscmat.h" I*/
#include <petsc/private/cudavecimpl.h> /* cublas definitions are here */

#if defined(PETSC_USE_COMPLEX)
#if defined(PETSC_USE_REAL_SINGLE)
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnCpotrf((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnCpotrf_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnCpotrs((a),(b),(c),(d),(hipComplex*)(e),(f),(hipComplex*)(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnCpotri((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnCpotri_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnCsytrf((a),(b),(c),(hipComplex*)(d),(e),(f),(hipComplex*)(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnCsytrf_bufferSize((a),(b),(hipComplex*)(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnCgetrf((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnCgetrf_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnCgetrs((a),(b),(c),(d),(hipComplex*)(e),(f),(g),(hipComplex*)(h),(i),(j))
#define hipsolverDnXgeqrf_bufferSize(a,b,c,d,e,f) hipsolverDnCgeqrf_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define hipsolverDnXgeqrf(a,b,c,d,e,f,g,h,i)      hipsolverDnCgeqrf((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(hipComplex*)(g),(h),(i))
#define cusolverDnXormqr_bufferSize(a,b,c,d,e,f,g,h,i,j,k,l) hipsolverDnCunmqr_bufferSize((a),(b),(c),(d),(e),(f),(hipComplex*)(g),(h),(hipComplex*)(i),(hipComplex*)(j),(k),(l))
#define cusolverDnXormqr(a,b,c,d,e,f,g,h,i,j,k,l,m,n) hipsolverDnCunmqr((a),(b),(c),(d),(e),(f),(hipComplex*)(g),(h),(hipComplex*)(i),(hipComplex*)(j),(k),(hipComplex*)(l),(m),(n))
#define cublasXtrsm(a,b,c,d,e,f,g,h,i,j,k,l)     hipblasCtrsm((a),(b),(c),(d),(e),(f),(g),(hipComplex*)(h),(hipComplex*)(i),(j),(hipComplex*)(k),(l))
#else /* complex double */
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnZpotrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnZpotrf_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnZpotrs((a),(b),(c),(d),(hipDoubleComplex*)(e),(f),(hipDoubleComplex*)(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnZpotri((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnZpotri_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnZsytrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(f),(hipDoubleComplex*)(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnZsytrf_bufferSize((a),(b),(hipDoubleComplex*)(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnZgetrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnZgetrf_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnZgetrs((a),(b),(c),(d),(hipDoubleComplex*)(e),(f),(g),(hipDoubleComplex*)(h),(i),(j))
#define hipsolverDnXgeqrf_bufferSize(a,b,c,d,e,f) hipsolverDnZgeqrf_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define hipsolverDnXgeqrf(a,b,c,d,e,f,g,h,i)      hipsolverDnZgeqrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(hipDoubleComplex*)(g),(h),(i))
#define cusolverDnXormqr_bufferSize(a,b,c,d,e,f,g,h,i,j,k,l) hipsolverDnZunmqr_bufferSize((a),(b),(c),(d),(e),(f),(hipDoubleComplex*)(g),(h),(hipDoubleComplex*)(i),(hipDoubleComplex*)(j),(k),(l))
#define cusolverDnXormqr(a,b,c,d,e,f,g,h,i,j,k,l,m,n) hipsolverDnZunmqr((a),(b),(c),(d),(e),(f),(hipDoubleComplex*)(g),(h),(hipDoubleComplex*)(i),(hipDoubleComplex*)(j),(k),(hipDoubleComplex*)(l),(m),(n))
#define cublasXtrsm(a,b,c,d,e,f,g,h,i,j,k,l)     hipblasZtrsm((a),(b),(c),(d),(e),(f),(g),(hipDoubleComplex*)(h),(hipDoubleComplex*)(i),(j),(hipDoubleComplex*)(k),(l))
#endif
#else /* real single */
#if defined(PETSC_USE_REAL_SINGLE)
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnSpotrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnSpotrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnSpotrs((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnSpotri((a),(b),(c),(d),(e),(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnSpotri_bufferSize((a),(b),(c),(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnSsytrf((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnSsytrf_bufferSize((a),(b),(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnSgetrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnSgetrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnSgetrs((a),(b),(c),(d),(e),(f),(g),(h),(i),(j))
#define hipsolverDnXgeqrf_bufferSize(a,b,c,d,e,f) hipsolverDnSgeqrf_bufferSize((a),(b),(c),(float*)(d),(e),(f))
#define hipsolverDnXgeqrf(a,b,c,d,e,f,g,h,i)      hipsolverDnSgeqrf((a),(b),(c),(float*)(d),(e),(float*)(f),(float*)(g),(h),(i))
#define cusolverDnXormqr_bufferSize(a,b,c,d,e,f,g,h,i,j,k,l) hipsolverDnSormqr_bufferSize((a),(b),(c),(d),(e),(f),(float*)(g),(h),(float*)(i),(float*)(j),(k),(l))
#define cusolverDnXormqr(a,b,c,d,e,f,g,h,i,j,k,l,m,n) hipsolverDnSormqr((a),(b),(c),(d),(e),(f),(float*)(g),(h),(float*)(i),(float*)(j),(k),(float*)(l),(m),(n))
#define cublasXtrsm(a,b,c,d,e,f,g,h,i,j,k,l)     hipblasStrsm((a),(b),(c),(d),(e),(f),(g),(float*)(h),(float*)(i),(j),(float*)(k),(l))
#else /* real double */
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnDpotrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnDpotrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnDpotrs((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnDpotri((a),(b),(c),(d),(e),(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnDpotri_bufferSize((a),(b),(c),(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnDsytrf((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnDsytrf_bufferSize((a),(b),(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnDgetrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnDgetrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnDgetrs((a),(b),(c),(d),(e),(f),(g),(h),(i),(j))
#define hipsolverDnXgeqrf_bufferSize(a,b,c,d,e,f) hipsolverDnDgeqrf_bufferSize((a),(b),(c),(double*)(d),(e),(f))
#define hipsolverDnXgeqrf(a,b,c,d,e,f,g,h,i)      hipsolverDnDgeqrf((a),(b),(c),(double*)(d),(e),(double*)(f),(double*)(g),(h),(i))
#define cusolverDnXormqr_bufferSize(a,b,c,d,e,f,g,h,i,j,k,l) hipsolverDnDormqr_bufferSize((a),(b),(c),(d),(e),(f),(double*)(g),(h),(double*)(i),(double*)(j),(k),(l))
#define cusolverDnXormqr(a,b,c,d,e,f,g,h,i,j,k,l,m,n) hipsolverDnDormqr((a),(b),(c),(d),(e),(f),(double*)(g),(h),(double*)(i),(double*)(j),(k),(double*)(l),(m),(n))
#define cublasXtrsm(a,b,c,d,e,f,g,h,i,j,k,l)     hipblasDtrsm((a),(b),(c),(d),(e),(f),(g),(double*)(h),(double*)(i),(j),(double*)(k),(l))
#endif
#endif

typedef struct {
  PetscScalar *d_v; /* pointer to the matrix on the GPU */
  PetscBool   user_alloc;
  PetscScalar *unplacedarray; /* if one called MatCUDADensePlaceArray(), this is where it stashed the original */
  PetscBool   unplaced_user_alloc;
  /* factorization support */
  PetscCuBLASInt *d_fact_ipiv; /* device pivots */
  PetscScalar *d_fact_tau;  /* device QR tau vector */
  PetscScalar *d_fact_work; /* device workspace */
  PetscCuBLASInt fact_lwork;
  PetscCuBLASInt *d_fact_info; /* device info */
  /* workspace */
  Vec         workvec;
} Mat_SeqDenseCUDA;

PetscErrorCode MatSeqDenseCUDASetPreallocation(Mat A, PetscScalar *d_data)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;
  PetscBool        iscuda;
  hipError_t      cerr;

  PetscFunctionBegin;
  ierr = PetscObjectTypeCompare((PetscObject)A,MATSEQDENSECUDA,&iscuda);CHKERRQ(ierr);
  if (!iscuda) PetscFunctionReturn(0);
  ierr = PetscLayoutSetUp(A->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->cmap);CHKERRQ(ierr);
  /* it may happen CPU preallocation has not been performed */
  if (cA->lda <= 0) cA->lda = A->rmap->n;
  if (!dA->user_alloc) { cerr = hipFree(dA->d_v);CHKERRCUDA(cerr); }
  if (!d_data) { /* petsc-allocated storage */
    size_t sz;

    ierr = PetscIntMultError(cA->lda,A->cmap->n,NULL);CHKERRQ(ierr);
    sz   = cA->lda*A->cmap->n*sizeof(PetscScalar);
    cerr = hipMalloc((void**)&dA->d_v,sz);CHKERRCUDA(cerr);
    cerr = hipMemset(dA->d_v,0,sz);CHKERRCUDA(cerr);
    dA->user_alloc = PETSC_FALSE;
  } else { /* user-allocated storage */
    dA->d_v        = d_data;
    dA->user_alloc = PETSC_TRUE;
  }
  A->offloadmask  = PETSC_OFFLOAD_GPU;
  A->preallocated = PETSC_TRUE;
  A->assembled    = PETSC_TRUE;
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqDenseCUDACopyFromGPU(Mat A)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;
  hipError_t      cerr;

  PetscFunctionBegin;
  PetscCheckTypeName(A,MATSEQDENSECUDA);
  ierr = PetscInfo(A,"%s matrix %d x %d\n",A->offloadmask == PETSC_OFFLOAD_GPU ? "Copy" : "Reusing",A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  if (A->offloadmask == PETSC_OFFLOAD_GPU) {
    if (!cA->v) { /* MatCreateSeqDenseCUDA may not allocate CPU memory. Allocate if needed */
      ierr = MatSeqDenseSetPreallocation(A,NULL);CHKERRQ(ierr);
    }
    ierr = PetscLogEventBegin(MAT_DenseCopyFromGPU,A,0,0,0);CHKERRQ(ierr);
    if (cA->lda > A->rmap->n) {
      cerr = hipMemcpy2D(cA->v,cA->lda*sizeof(PetscScalar),dA->d_v,cA->lda*sizeof(PetscScalar),A->rmap->n*sizeof(PetscScalar),A->cmap->n,hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    } else {
      cerr = hipMemcpy(cA->v,dA->d_v,cA->lda*sizeof(PetscScalar)*A->cmap->n,hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    }
    ierr = PetscLogGpuToCpu(cA->lda*sizeof(PetscScalar)*A->cmap->n);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(MAT_DenseCopyFromGPU,A,0,0,0);CHKERRQ(ierr);

    A->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqDenseCUDACopyToGPU(Mat A)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscBool        copy;
  PetscErrorCode   ierr;
  hipError_t      cerr;

  PetscFunctionBegin;
  PetscCheckTypeName(A,MATSEQDENSECUDA);
  if (A->boundtocpu) PetscFunctionReturn(0);
  copy = (PetscBool)(A->offloadmask == PETSC_OFFLOAD_CPU || A->offloadmask == PETSC_OFFLOAD_UNALLOCATED);
  ierr = PetscInfo(A,"%s matrix %d x %d\n",copy ? "Copy" : "Reusing",A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  if (copy) {
    if (!dA->d_v) { /* Allocate GPU memory if not present */
      ierr = MatSeqDenseCUDASetPreallocation(A,NULL);CHKERRQ(ierr);
    }
    ierr = PetscLogEventBegin(MAT_DenseCopyToGPU,A,0,0,0);CHKERRQ(ierr);
    if (cA->lda > A->rmap->n) {
      cerr = hipMemcpy2D(dA->d_v,cA->lda*sizeof(PetscScalar),cA->v,cA->lda*sizeof(PetscScalar),A->rmap->n*sizeof(PetscScalar),A->cmap->n,hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    } else {
      cerr = hipMemcpy(dA->d_v,cA->v,cA->lda*sizeof(PetscScalar)*A->cmap->n,hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    }
    ierr = PetscLogCpuToGpu(cA->lda*sizeof(PetscScalar)*A->cmap->n);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(MAT_DenseCopyToGPU,A,0,0,0);CHKERRQ(ierr);

    A->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCopy_SeqDenseCUDA(Mat A,Mat B,MatStructure str)
{
  PetscErrorCode    ierr;
  const PetscScalar *va;
  PetscScalar       *vb;
  PetscInt          lda1,lda2,m=A->rmap->n,n=A->cmap->n;
  hipError_t       cerr;

  PetscFunctionBegin;
  /* If the two matrices don't have the same copy implementation, they aren't compatible for fast copy. */
  if (A->ops->copy != B->ops->copy) {
    ierr = MatCopy_Basic(A,B,str);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }
  PetscCheckFalse(m != B->rmap->n || n != B->cmap->n,PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"size(B) != size(A)");
  ierr = MatDenseCUDAGetArrayRead(A,&va);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayWrite(B,&vb);CHKERRQ(ierr);
  ierr = MatDenseGetLDA(A,&lda1);CHKERRQ(ierr);
  ierr = MatDenseGetLDA(B,&lda2);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (lda1>m || lda2>m) {
    cerr = hipMemcpy2D(vb,lda2*sizeof(PetscScalar),va,lda1*sizeof(PetscScalar),m*sizeof(PetscScalar),n,hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
  } else {
    cerr = hipMemcpy(vb,va,m*(n*sizeof(PetscScalar)),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
  }
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayWrite(B,&vb);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&va);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatZeroEntries_SeqDenseCUDA(Mat A)
{
  PetscErrorCode ierr;
  PetscScalar    *va;
  PetscInt       lda,m = A->rmap->n,n = A->cmap->n;
  hipError_t    cerr;

  PetscFunctionBegin;
  ierr = MatDenseCUDAGetArrayWrite(A,&va);CHKERRQ(ierr);
  ierr = MatDenseGetLDA(A,&lda);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (lda>m) {
    cerr = hipMemset2D(va,lda*sizeof(PetscScalar),0,m*sizeof(PetscScalar),n);CHKERRCUDA(cerr);
  } else {
    cerr = hipMemset(va,0,m*(n*sizeof(PetscScalar)));CHKERRCUDA(cerr);
  }
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayWrite(A,&va);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAPlaceArray_SeqDenseCUDA(Mat A, const PetscScalar *a)
{
  Mat_SeqDense     *aa = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  PetscCheckFalse(aa->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheckFalse(aa->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  PetscCheckFalse(dA->unplacedarray,PETSC_COMM_SELF,PETSC_ERR_ORDER,"MatDenseCUDAResetArray() must be called first");
  if (aa->v) { ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr); }
  dA->unplacedarray = dA->d_v;
  dA->unplaced_user_alloc = dA->user_alloc;
  dA->d_v = (PetscScalar*)a;
  dA->user_alloc = PETSC_TRUE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAResetArray_SeqDenseCUDA(Mat A)
{
  Mat_SeqDense     *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  PetscCheckFalse(a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheckFalse(a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  if (a->v) { ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr); }
  dA->d_v = dA->unplacedarray;
  dA->user_alloc = dA->unplaced_user_alloc;
  dA->unplacedarray = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAReplaceArray_SeqDenseCUDA(Mat A, const PetscScalar *a)
{
  Mat_SeqDense     *aa = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  hipError_t      cerr;

  PetscFunctionBegin;
  PetscCheckFalse(aa->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheckFalse(aa->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  PetscCheckFalse(dA->unplacedarray,PETSC_COMM_SELF,PETSC_ERR_ORDER,"MatDenseCUDAResetArray() must be called first");
  if (!dA->user_alloc) { cerr = hipFree(dA->d_v);CHKERRCUDA(cerr); }
  dA->d_v = (PetscScalar*)a;
  dA->user_alloc = PETSC_FALSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAGetArrayWrite_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (!dA->d_v) {
    ierr = MatSeqDenseCUDASetPreallocation(A,NULL);CHKERRQ(ierr);
  }
  *a = dA->d_v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDARestoreArrayWrite_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  PetscFunctionBegin;
  if (a) *a = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAGetArrayRead_SeqDenseCUDA(Mat A, const PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr);
  *a   = dA->d_v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDARestoreArrayRead_SeqDenseCUDA(Mat A, const PetscScalar **a)
{
  PetscFunctionBegin;
  if (a) *a = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAGetArray_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr);
  *a   = dA->d_v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDARestoreArray_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  PetscFunctionBegin;
  if (a) *a = NULL;
  PetscFunctionReturn(0);
}

PETSC_EXTERN PetscErrorCode MatSeqDenseCUDAInvertFactors_Private(Mat A)
{
#if PETSC_PKG_CUDA_VERSION_GE(10,1,0)
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  PetscErrorCode     ierr;
  hipError_t        ccer;
  hipsolverStatus_t   cerr;
  hipsolverHandle_t handle;
  PetscCuBLASInt     n,lda;
#if defined(PETSC_USE_DEBUG)
  PetscCuBLASInt     info;
#endif

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(A->cmap->n,&n);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(a->lda,&lda);CHKERRQ(ierr);
  PetscCheckFalse(A->factortype == MAT_FACTOR_LU,PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDngetri not implemented");
  if (A->factortype == MAT_FACTOR_CHOLESKY) {
    if (!dA->d_fact_ipiv) { /* spd */
      PetscCuBLASInt il;

      ierr = MatDenseCUDAGetArray(A,&da);CHKERRQ(ierr);
      cerr = cusolverDnXpotri_bufferSize(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,&il);CHKERRCUSOLVER(cerr);
      if (il > dA->fact_lwork) {
        dA->fact_lwork = il;

        ccer = hipFree(dA->d_fact_work);CHKERRCUDA(ccer);
        ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
      }
      ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
      cerr = cusolverDnXpotri(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info);CHKERRCUSOLVER(cerr);
      ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
      ierr = MatDenseCUDARestoreArray(A,&da);CHKERRQ(ierr);
      /* TODO (write cuda kernel) */
      ierr = MatSeqDenseSymmetrize_Private(A,PETSC_TRUE);CHKERRQ(ierr);
    } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDnsytri not implemented");
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"Not implemented");
#if defined(PETSC_USE_DEBUG)
  ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
  PetscCheckFalse(info > 0,PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: leading minor of order %d is zero",info);
  PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
  ierr = PetscLogGpuFlops(1.0*n*n*n/3.0);CHKERRQ(ierr);
  A->ops->solve          = NULL;
  A->ops->solvetranspose = NULL;
  A->ops->matsolve       = NULL;
  A->factortype          = MAT_FACTOR_NONE;

  ierr = PetscFree(A->solvertype);CHKERRQ(ierr);
  PetscFunctionReturn(0);
#else
  SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Upgrade to CUDA version 10.1.0 or higher");
#endif
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Internal(Mat A, Vec xx, Vec yy, PetscBool transpose,
                                                     PetscErrorCode (*matsolve)(Mat,PetscScalar*,PetscCuBLASInt,PetscCuBLASInt,PetscCuBLASInt,PetscCuBLASInt,PetscBool))
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar      *y;
  PetscCuBLASInt   m=0, k=0;
  PetscBool        xiscuda, yiscuda, aiscuda;
  hipError_t      cerr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  PetscCheckFalse(A->factortype == MAT_FACTOR_NONE,PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  ierr = PetscCuBLASIntCast(A->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(A->cmap->n,&k);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)xx,VECSEQCUDA,&xiscuda);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)yy,VECSEQCUDA,&yiscuda);CHKERRQ(ierr);
  {
    const PetscScalar *x;
    PetscBool xishost = PETSC_TRUE;

    /* The logic here is to try to minimize the amount of memory copying:
       if we call VecCUDAGetArrayRead(X,&x) every time xiscuda and the
       data is not offloaded to the GPU yet, then the data is copied to the
       GPU.  But we are only trying to get the data in order to copy it into the y
       array.  So the array x will be wherever the data already is so that
       only one memcpy is performed */
    if (xiscuda && xx->offloadmask & PETSC_OFFLOAD_GPU) {
      ierr = VecCUDAGetArrayRead(xx, &x);CHKERRQ(ierr);
      xishost =  PETSC_FALSE;
    } else {
      ierr = VecGetArrayRead(xx, &x);CHKERRQ(ierr);
    }
    if (k < m || !yiscuda) {
      if (!dA->workvec) {
        ierr = VecCreateSeqCUDA(PetscObjectComm((PetscObject)A), m, &(dA->workvec));CHKERRQ(ierr);
      }
      ierr = VecCUDAGetArrayWrite(dA->workvec, &y);CHKERRQ(ierr);
    } else {
      ierr = VecCUDAGetArrayWrite(yy,&y);CHKERRQ(ierr);
    }
    cerr = hipMemcpy(y,x,m*sizeof(PetscScalar),xishost ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
  }
  ierr = PetscObjectTypeCompare((PetscObject)A,MATSEQDENSECUDA,&aiscuda);CHKERRQ(ierr);
  if (!aiscuda) {
    ierr = MatConvert(A,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&A);CHKERRQ(ierr);
  }
  ierr = (*matsolve) (A, y, m, m, 1, k, transpose);CHKERRQ(ierr);
  if (!aiscuda) {
    ierr = MatConvert(A,MATSEQDENSE,MAT_INPLACE_MATRIX,&A);CHKERRQ(ierr);
  }
  if (k < m || !yiscuda) {
    PetscScalar *yv;

    /* The logic here is that the data is not yet in either yy's GPU array or its
       CPU array.  There is nothing in the interface to say where the user would like
       it to end up.  So we choose the GPU, because it is the faster option */
    if (yiscuda) {
      ierr = VecCUDAGetArrayWrite(yy,&yv);CHKERRQ(ierr);
    } else {
      ierr = VecGetArray(yy,&yv);CHKERRQ(ierr);
    }
    cerr = hipMemcpy(yv,y,k*sizeof(PetscScalar),yiscuda ? hipMemcpyDeviceToDevice: hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    if (yiscuda) {
      ierr = VecCUDARestoreArrayWrite(yy,&yv);CHKERRQ(ierr);
    } else {
      ierr = VecRestoreArray(yy,&yv);CHKERRQ(ierr);
    }
    ierr = VecCUDARestoreArrayWrite(dA->workvec, &y);CHKERRQ(ierr);
  } else {
    ierr = VecCUDARestoreArrayWrite(yy,&y);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolve_SeqDenseCUDA_Internal(Mat A, Mat B, Mat X, PetscBool transpose,
                                                        PetscErrorCode (*matsolve)(Mat,PetscScalar*,PetscCuBLASInt,PetscCuBLASInt,PetscCuBLASInt,PetscCuBLASInt,PetscBool))
{
  PetscScalar       *y;
  PetscInt          n, _ldb, _ldx;
  PetscBool         biscuda, xiscuda, aiscuda;
  PetscCuBLASInt    nrhs=0,m=0,k=0,ldb=0,ldx=0,ldy=0;
  hipError_t       cerr;
  PetscErrorCode    ierr;

  PetscFunctionBegin;
  PetscCheckFalse(A->factortype == MAT_FACTOR_NONE,PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  ierr = PetscCuBLASIntCast(A->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(A->cmap->n,&k);CHKERRQ(ierr);
  ierr = MatGetSize(B,NULL,&n);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(n,&nrhs);CHKERRQ(ierr);
  ierr = MatDenseGetLDA(B,&_ldb);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(_ldb, &ldb);CHKERRQ(ierr);
  ierr = MatDenseGetLDA(X,&_ldx);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(_ldx, &ldx);CHKERRQ(ierr);

  ierr = PetscObjectTypeCompare((PetscObject)B,MATSEQDENSECUDA,&biscuda);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)X,MATSEQDENSECUDA,&xiscuda);CHKERRQ(ierr);
  {
    /* The logic here is to try to minimize the amount of memory copying:
       if we call MatDenseCUDAGetArrayRead(B,&b) every time biscuda and the
       data is not offloaded to the GPU yet, then the data is copied to the
       GPU.  But we are only trying to get the data in order to copy it into the y
       array.  So the array b will be wherever the data already is so that
       only one memcpy is performed */
    const PetscScalar *b;

    /* some copying from B will be involved */
    PetscBool bishost = PETSC_TRUE;

    if (biscuda && B->offloadmask & PETSC_OFFLOAD_GPU) {
      ierr = MatDenseCUDAGetArrayRead(B,&b);CHKERRQ(ierr);
      bishost = PETSC_FALSE;
    } else {
      ierr = MatDenseGetArrayRead(B,&b);CHKERRQ(ierr);
    }
    if (ldx < m || !xiscuda) {
      /* X's array cannot serve as the array (too small or not on device), B's
       * array cannot serve as the array (const), so allocate a new array  */
      ldy = m;
      cerr = hipMalloc((void**)&y,nrhs*m*sizeof(PetscScalar));CHKERRCUDA(cerr);
    } else {
      /* X's array should serve as the array */
      ldy = ldx;
      ierr = MatDenseCUDAGetArrayWrite(X,&y);CHKERRQ(ierr);
    }
    cerr = hipMemcpy2D(y,ldy*sizeof(PetscScalar),b,ldb*sizeof(PetscScalar),m*sizeof(PetscScalar),nrhs,bishost ? hipMemcpyHostToDevice: hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
    if (bishost) {
      ierr = MatDenseRestoreArrayRead(B,&b);CHKERRQ(ierr);
    } else {
      ierr = MatDenseCUDARestoreArrayRead(B,&b);CHKERRQ(ierr);
    }
  }
  ierr = PetscObjectTypeCompare((PetscObject)A,MATSEQDENSECUDA,&aiscuda);CHKERRQ(ierr);
  if (!aiscuda) {
    ierr = MatConvert(A,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&A);CHKERRQ(ierr);
  }
  ierr = (*matsolve) (A, y, ldy, m, nrhs, k, transpose);CHKERRQ(ierr);
  if (!aiscuda) {
    ierr = MatConvert(A,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&A);CHKERRQ(ierr);
  }
  if (ldx < m || !xiscuda) {
    PetscScalar *x;

    /* The logic here is that the data is not yet in either X's GPU array or its
       CPU array.  There is nothing in the interface to say where the user would like
       it to end up.  So we choose the GPU, because it is the faster option */
    if (xiscuda) {
      ierr = MatDenseCUDAGetArrayWrite(X,&x);CHKERRQ(ierr);
    } else {
      ierr = MatDenseGetArray(X,&x);CHKERRQ(ierr);
    }
    cerr = hipMemcpy2D(x,ldx*sizeof(PetscScalar),y,ldy*sizeof(PetscScalar),k*sizeof(PetscScalar),nrhs,xiscuda ? hipMemcpyDeviceToDevice: hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    if (xiscuda) {
      ierr = MatDenseCUDARestoreArrayWrite(X,&x);CHKERRQ(ierr);
    } else {
      ierr = MatDenseRestoreArray(X,&x);CHKERRQ(ierr);
    }
    cerr = hipFree(y);CHKERRCUDA(cerr);
  } else {
    ierr = MatDenseCUDARestoreArrayWrite(X,&y);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Internal_LU(Mat A, PetscScalar *x, PetscCuBLASInt ldx, PetscCuBLASInt m, PetscCuBLASInt nrhs, PetscCuBLASInt k, PetscBool T)
{
  Mat_SeqDense       *mat = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  const PetscScalar  *da;
  PetscCuBLASInt     lda;
  hipsolverHandle_t handle;
  hipError_t        ccer;
  hipsolverStatus_t   cerr;
  int                info;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(mat->lda,&lda);CHKERRQ(ierr);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  ierr = PetscInfo(A,"LU solve %d x %d on backend\n",m,k);CHKERRQ(ierr);
  cerr = hipsolverDnXgetrs(handle,T ? HIPBLAS_OP_T : HIPBLAS_OP_N,m,nrhs,da,lda,dA->d_fact_ipiv,x,ldx,dA->d_fact_info);CHKERRCUSOLVER(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  if (PetscDefined(USE_DEBUG)) {
    ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
    PetscCheckFalse(info > 0,PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
    else PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
  }
  ierr = PetscLogGpuFlops(nrhs*(2.0*m*m - m));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Internal_Cholesky(Mat A, PetscScalar *x, PetscCuBLASInt ldx, PetscCuBLASInt m, PetscCuBLASInt nrhs, PetscCuBLASInt k, PetscBool T)
{
  Mat_SeqDense       *mat = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  const PetscScalar  *da;
  PetscCuBLASInt     lda;
  hipsolverHandle_t handle;
  hipError_t        ccer;
  hipsolverStatus_t   cerr;
  int                info;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(mat->lda,&lda);CHKERRQ(ierr);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  ierr = PetscInfo(A,"Cholesky solve %d x %d on backend\n",m,k);CHKERRQ(ierr);
  if (!dA->d_fact_ipiv) { /* spd */
    /* ========= Program hit hipErrorNotReady (error 34) due to "device not ready" on CUDA API call to hipEventQuery. */
    cerr = hipsolverDnXpotrs(handle,HIPBLAS_FILL_MODE_LOWER,m,nrhs,da,lda,x,ldx,dA->d_fact_info);CHKERRCUSOLVER(cerr);
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDnsytrs not implemented");
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  if (PetscDefined(USE_DEBUG)) {
    ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
    PetscCheckFalse(info > 0,PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
    else PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
  }
  ierr = PetscLogGpuFlops(nrhs*(2.0*m*m - m));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Internal_QR(Mat A, PetscScalar *x, PetscCuBLASInt ldx, PetscCuBLASInt m, PetscCuBLASInt nrhs, PetscCuBLASInt k, PetscBool T)
{
  Mat_SeqDense       *mat = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  const PetscScalar  *da;
  PetscCuBLASInt     lda, rank;
  hipsolverHandle_t handle;
  hipblasHandle_t     bhandle;
  hipError_t        ccer;
  hipsolverStatus_t   csrr;
  hipblasStatus_t     cbrr;
  int                info;
  hipblasOperation_t  trans;
  PetscScalar        one = 1.;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = PetscCuBLASIntCast(mat->rank,&rank);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(mat->lda,&lda);CHKERRQ(ierr);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&bhandle);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  ierr = PetscInfo(A,"QR solve %d x %d on backend\n",m,k);CHKERRQ(ierr);
  if (!T) {
    if (PetscDefined(USE_COMPLEX)) {
      trans = HIPBLAS_OP_C;
    } else {
      trans = HIPBLAS_OP_T;
    }
    csrr = cusolverDnXormqr(handle, HIPBLAS_SIDE_LEFT, trans, m, nrhs, rank, da, lda, dA->d_fact_tau, x, ldx, dA->d_fact_work, dA->fact_lwork, dA->d_fact_info);CHKERRCUSOLVER(csrr);
    if (PetscDefined(USE_DEBUG)) {
      ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
      PetscCheckFalse(info != 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
    }
    cbrr = cublasXtrsm(bhandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, rank, nrhs, &one, da, lda, x, ldx);CHKERRCUBLAS(cbrr);
  } else {
    cbrr = cublasXtrsm(bhandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, rank, nrhs, &one, da, lda, x, ldx);CHKERRCUBLAS(cbrr);
    csrr = cusolverDnXormqr(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_N, m, nrhs, rank, da, lda, dA->d_fact_tau, x, ldx, dA->d_fact_work, dA->fact_lwork, dA->d_fact_info);CHKERRCUSOLVER(csrr);
    if (PetscDefined(USE_DEBUG)) {
      ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
      PetscCheckFalse(info != 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
    }
  }
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  ierr = PetscLogFlops(nrhs*(4.0*m*mat->rank - PetscSqr(mat->rank)));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_LU(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_LU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqDenseCUDA_LU(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_LU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Cholesky(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_Cholesky);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqDenseCUDA_Cholesky(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_Cholesky);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_QR(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_QR);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqDenseCUDA_QR(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_QR);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolve_SeqDenseCUDA_LU(Mat A,Mat B,Mat X)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_LU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolveTranspose_SeqDenseCUDA_LU(Mat A,Mat B,Mat X)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_LU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolve_SeqDenseCUDA_Cholesky(Mat A,Mat B,Mat X)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_Cholesky);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolveTranspose_SeqDenseCUDA_Cholesky(Mat A,Mat B,Mat X)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_Cholesky);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolve_SeqDenseCUDA_QR(Mat A,Mat B,Mat X)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_QR);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolveTranspose_SeqDenseCUDA_QR(Mat A,Mat B,Mat X)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_QR);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatLUFactor_SeqDenseCUDA(Mat A,IS rperm,IS cperm,const MatFactorInfo *factinfo)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  PetscCuBLASInt     m,n,lda;
#if defined(PETSC_USE_DEBUG)
  int                info;
#endif
  hipsolverStatus_t   cerr;
  hipsolverHandle_t handle;
  hipError_t        ccer;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArray(A,&da);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(A->cmap->n,&n);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(A->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(a->lda,&lda);CHKERRQ(ierr);
  ierr = PetscInfo(A,"LU factor %d x %d on backend\n",m,n);CHKERRQ(ierr);
  if (!dA->d_fact_ipiv) {
    ccer = hipMalloc((void**)&dA->d_fact_ipiv,n*sizeof(*dA->d_fact_ipiv));CHKERRCUDA(ccer);
  }
  if (!dA->fact_lwork) {
    cerr = hipsolverDnXgetrf_bufferSize(handle,m,n,da,lda,&dA->fact_lwork);CHKERRCUSOLVER(cerr);
    ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
  }
  if (!dA->d_fact_info) {
    ccer = hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info));CHKERRCUDA(ccer);
  }
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  cerr = hipsolverDnXgetrf(handle,m,n,da,lda,dA->d_fact_work,dA->d_fact_ipiv,dA->d_fact_info);CHKERRCUSOLVER(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArray(A,&da);CHKERRQ(ierr);
#if defined(PETSC_USE_DEBUG)
  ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
  PetscCheckFalse(info > 0,PETSC_COMM_SELF,PETSC_ERR_MAT_LU_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
  else PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
  A->factortype = MAT_FACTOR_LU;
  ierr = PetscLogGpuFlops(2.0*n*n*m/3.0);CHKERRQ(ierr);

  A->ops->solve             = MatSolve_SeqDenseCUDA_LU;
  A->ops->solvetranspose    = MatSolveTranspose_SeqDenseCUDA_LU;
  A->ops->matsolve          = MatMatSolve_SeqDenseCUDA_LU;
  A->ops->matsolvetranspose = MatMatSolveTranspose_SeqDenseCUDA_LU;

  ierr = PetscFree(A->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATSOLVERCUDA,&A->solvertype);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCholeskyFactor_SeqDenseCUDA(Mat A,IS perm,const MatFactorInfo *factinfo)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  PetscCuBLASInt     n,lda;
#if defined(PETSC_USE_DEBUG)
  int                info;
#endif
  hipsolverStatus_t   cerr;
  hipsolverHandle_t handle;
  hipError_t        ccer;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(A->rmap->n,&n);CHKERRQ(ierr);
  ierr = PetscInfo(A,"Cholesky factor %d x %d on backend\n",n,n);CHKERRQ(ierr);
  if (A->spd) {
    ierr = MatDenseCUDAGetArray(A,&da);CHKERRQ(ierr);
    ierr = PetscCuBLASIntCast(a->lda,&lda);CHKERRQ(ierr);
    if (!dA->fact_lwork) {
      cerr = hipsolverDnXpotrf_bufferSize(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,&dA->fact_lwork);CHKERRCUSOLVER(cerr);
      ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
    }
    if (!dA->d_fact_info) {
      ccer = hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info));CHKERRCUDA(ccer);
    }
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cerr = hipsolverDnXpotrf(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info);CHKERRCUSOLVER(cerr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

    ierr = MatDenseCUDARestoreArray(A,&da);CHKERRQ(ierr);
#if defined(PETSC_USE_DEBUG)
    ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
    PetscCheckFalse(info > 0,PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
    else PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
    A->factortype = MAT_FACTOR_CHOLESKY;
    ierr = PetscLogGpuFlops(1.0*n*n*n/3.0);CHKERRQ(ierr);
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"cusolverDnsytrs unavailable. Use MAT_FACTOR_LU");
#if 0
    /* at the time of writing this interface (cuda 10.0), cusolverDn does not implement *sytrs and *hetr* routines
       The code below should work, and it can be activated when *sytrs routines will be available */
    if (!dA->d_fact_ipiv) {
      ccer = hipMalloc((void**)&dA->d_fact_ipiv,n*sizeof(*dA->d_fact_ipiv));CHKERRCUDA(ccer);
    }
    if (!dA->fact_lwork) {
      cerr = cusolverDnXsytrf_bufferSize(handle,n,da,lda,&dA->fact_lwork);CHKERRCUSOLVER(cerr);
      ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
    }
    if (!dA->d_fact_info) {
      ccer = hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info));CHKERRCUDA(ccer);
    }
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cerr = cusolverDnXsytrf(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,dA->d_fact_ipiv,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info);CHKERRCUSOLVER(cerr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
#endif

  A->ops->solve             = MatSolve_SeqDenseCUDA_Cholesky;
  A->ops->solvetranspose    = MatSolveTranspose_SeqDenseCUDA_Cholesky;
  A->ops->matsolve          = MatMatSolve_SeqDenseCUDA_Cholesky;
  A->ops->matsolvetranspose = MatMatSolveTranspose_SeqDenseCUDA_Cholesky;
  ierr = PetscFree(A->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATSOLVERCUDA,&A->solvertype);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatQRFactor_SeqDenseCUDA(Mat A,IS col,const MatFactorInfo *factinfo)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  PetscCuBLASInt     m,min,max,n,lda;
#if defined(PETSC_USE_DEBUG)
  int                info;
#endif
  hipsolverStatus_t   cerr;
  hipsolverHandle_t handle;
  hipError_t        ccer;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArray(A,&da);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(A->cmap->n,&n);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(A->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(a->lda,&lda);CHKERRQ(ierr);
  ierr = PetscInfo(A,"QR factor %d x %d on backend\n",m,n);CHKERRQ(ierr);
  max = PetscMax(m,n);
  min = PetscMin(m,n);
  if (!dA->d_fact_tau) {
    ccer = hipMalloc((void**)&dA->d_fact_tau,min*sizeof(*dA->d_fact_tau));CHKERRCUDA(ccer);
  }
  if (!dA->d_fact_ipiv) {
    ccer = hipMalloc((void**)&dA->d_fact_ipiv,n*sizeof(*dA->d_fact_ipiv));CHKERRCUDA(ccer);
  }
  if (!dA->fact_lwork) {
    cerr = hipsolverDnXgeqrf_bufferSize(handle,m,n,da,lda,&dA->fact_lwork);CHKERRCUSOLVER(cerr);
    ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
  }
  if (!dA->d_fact_info) {
    ccer = hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info));CHKERRCUDA(ccer);
  }
  if (!dA->workvec) {
    ierr = VecCreateSeqCUDA(PetscObjectComm((PetscObject)A), m, &(dA->workvec));CHKERRQ(ierr);
  }
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  cerr = hipsolverDnXgeqrf(handle,m,n,da,lda,dA->d_fact_tau,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info);CHKERRCUSOLVER(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArray(A,&da);CHKERRQ(ierr);
#if defined(PETSC_USE_DEBUG)
  ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
  PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
  A->factortype = MAT_FACTOR_QR;
  a->rank = min;
  ierr = PetscLogGpuFlops(2.0*min*min*(max-min/3.0));CHKERRQ(ierr);

  A->ops->solve             = MatSolve_SeqDenseCUDA_QR;
  A->ops->solvetranspose    = MatSolveTranspose_SeqDenseCUDA_QR;
  A->ops->matsolve          = MatMatSolve_SeqDenseCUDA_QR;
  A->ops->matsolvetranspose = MatMatSolveTranspose_SeqDenseCUDA_QR;

  ierr = PetscFree(A->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATSOLVERCUDA,&A->solvertype);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* GEMM kernel: C = op(A)*op(B), tA, tB flag transposition */
PETSC_INTERN PetscErrorCode MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(Mat A,Mat B,Mat C,PetscBool tA,PetscBool tB)
{
  const PetscScalar *da,*db;
  PetscScalar       *dc;
  PetscScalar       one=1.0,zero=0.0;
  PetscCuBLASInt    m,n,k;
  PetscInt          alda,blda,clda;
  PetscErrorCode    ierr;
  hipblasHandle_t    cublasv2handle;
  PetscBool         Aiscuda,Biscuda;
  hipblasStatus_t    berr;

  PetscFunctionBegin;
  /* we may end up with SEQDENSE as one of the arguments */
  ierr = PetscObjectTypeCompare((PetscObject)A,MATSEQDENSECUDA,&Aiscuda);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)B,MATSEQDENSECUDA,&Biscuda);CHKERRQ(ierr);
  if (!Aiscuda) {
    ierr = MatConvert(A,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&A);CHKERRQ(ierr);
  }
  if (!Biscuda) {
    ierr = MatConvert(B,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&B);CHKERRQ(ierr);
  }
  ierr = PetscCuBLASIntCast(C->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(C->cmap->n,&n);CHKERRQ(ierr);
  if (tA) {
    ierr = PetscCuBLASIntCast(A->rmap->n,&k);CHKERRQ(ierr);
  } else {
    ierr = PetscCuBLASIntCast(A->cmap->n,&k);CHKERRQ(ierr);
  }
  if (!m || !n || !k) PetscFunctionReturn(0);
  ierr = PetscInfo(C,"Matrix-Matrix product %d x %d x %d on backend\n",m,k,n);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(B,&db);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayWrite(C,&dc);CHKERRQ(ierr);
  ierr = MatDenseGetLDA(A,&alda);CHKERRQ(ierr);
  ierr = MatDenseGetLDA(B,&blda);CHKERRQ(ierr);
  ierr = MatDenseGetLDA(C,&clda);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  berr = cublasXgemm(cublasv2handle,tA ? HIPBLAS_OP_T : HIPBLAS_OP_N,tB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                     m,n,k,&one,da,alda,db,blda,&zero,dc,clda);CHKERRCUBLAS(berr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(1.0*m*n*k + 1.0*m*n*(k-1));CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(B,&db);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayWrite(C,&dc);CHKERRQ(ierr);
  if (!Aiscuda) {
    ierr = MatConvert(A,MATSEQDENSE,MAT_INPLACE_MATRIX,&A);CHKERRQ(ierr);
  }
  if (!Biscuda) {
    ierr = MatConvert(B,MATSEQDENSE,MAT_INPLACE_MATRIX,&B);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatTransposeMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_TRUE,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_FALSE,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMatTransposeMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_FALSE,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatProductSetFromOptions_SeqDenseCUDA(Mat C)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatProductSetFromOptions_SeqDense(C);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* zz = op(A)*xx + yy
   if yy == NULL, only MatMult */
static PetscErrorCode MatMultAdd_SeqDenseCUDA_Private(Mat A,Vec xx,Vec yy,Vec zz,PetscBool trans)
{
  Mat_SeqDense      *mat = (Mat_SeqDense*)A->data;
  const PetscScalar *xarray,*da;
  PetscScalar       *zarray;
  PetscScalar       one=1.0,zero=0.0;
  PetscCuBLASInt    m, n, lda;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    berr;
  PetscErrorCode    ierr;

  PetscFunctionBegin;
  if (yy && yy != zz) { /* mult add */
    ierr = VecCopy_SeqCUDA(yy,zz);CHKERRQ(ierr);
  }
  if (!A->rmap->n || !A->cmap->n) {
    if (!yy) { /* mult only */
      ierr = VecSet_SeqCUDA(zz,0.0);CHKERRQ(ierr);
    }
    PetscFunctionReturn(0);
  }
  ierr = PetscInfo(A,"Matrix-vector product %d x %d on backend\n",A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(A->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(A->cmap->n,&n);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(mat->lda,&lda);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArray(zz,&zarray);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  berr = cublasXgemv(cublasv2handle,trans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                     m,n,&one,da,lda,xarray,1,(yy ? &one : &zero),zarray,1);CHKERRCUBLAS(berr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*A->rmap->n*A->cmap->n - (yy ? 0 : A->rmap->n));CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArray(zz,&zarray);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultAdd_SeqDenseCUDA(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,yy,zz,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultTransposeAdd_SeqDenseCUDA(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,yy,zz,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMult_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,NULL,yy,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultTranspose_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,NULL,yy,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetArrayRead_SeqDenseCUDA(Mat A,const PetscScalar **array)
{
  Mat_SeqDense   *mat = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyFromGPU(A);CHKERRQ(ierr);
  *array = mat->v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetArrayWrite_SeqDenseCUDA(Mat A,PetscScalar **array)
{
  Mat_SeqDense   *mat = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (!mat->v) { /* MatCreateSeqDenseCUDA may not allocate CPU memory. Allocate if needed */
    ierr = MatSeqDenseSetPreallocation(A,NULL);CHKERRQ(ierr);
  }
  *array = mat->v;
  A->offloadmask = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetArray_SeqDenseCUDA(Mat A,PetscScalar **array)
{
  Mat_SeqDense   *mat = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyFromGPU(A);CHKERRQ(ierr);
  *array = mat->v;
  A->offloadmask = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

PetscErrorCode MatScale_SeqDenseCUDA(Mat Y,PetscScalar alpha)
{
  Mat_SeqDense   *y = (Mat_SeqDense*)Y->data;
  PetscScalar    *dy;
  PetscCuBLASInt j,N,m,lday,one = 1;
  hipblasHandle_t cublasv2handle;
  hipblasStatus_t berr;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArray(Y,&dy);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(Y->rmap->n*Y->cmap->n,&N);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(Y->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(y->lda,&lday);CHKERRQ(ierr);
  ierr = PetscInfo(Y,"Performing Scale %d x %d on backend\n",Y->rmap->n,Y->cmap->n);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (lday>m) {
    for (j=0; j<Y->cmap->n; j++) {
      berr = cublasXscal(cublasv2handle,m,&alpha,dy+lday*j,one);CHKERRCUBLAS(berr);
    }
  } else {
    berr = cublasXscal(cublasv2handle,N,&alpha,dy,one);CHKERRCUBLAS(berr);
  }
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(N);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArray(Y,&dy);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatAXPY_SeqDenseCUDA(Mat Y,PetscScalar alpha,Mat X,MatStructure str)
{
  Mat_SeqDense      *x = (Mat_SeqDense*)X->data;
  Mat_SeqDense      *y = (Mat_SeqDense*)Y->data;
  const PetscScalar *dx;
  PetscScalar       *dy;
  PetscCuBLASInt    j,N,m,ldax,lday,one = 1;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    berr;
  PetscErrorCode    ierr;

  PetscFunctionBegin;
  if (!X->rmap->n || !X->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(X,&dx);CHKERRQ(ierr);
  if (alpha != 0.0) {
    ierr = MatDenseCUDAGetArray(Y,&dy);CHKERRQ(ierr);
  } else {
    ierr = MatDenseCUDAGetArrayWrite(Y,&dy);CHKERRQ(ierr);
  }
  ierr = PetscCuBLASIntCast(X->rmap->n*X->cmap->n,&N);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(X->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(x->lda,&ldax);CHKERRQ(ierr);
  ierr = PetscCuBLASIntCast(y->lda,&lday);CHKERRQ(ierr);
  ierr = PetscInfo(Y,"Performing AXPY %d x %d on backend\n",Y->rmap->n,Y->cmap->n);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (ldax>m || lday>m) {
    for (j=0; j<X->cmap->n; j++) {
      berr = cublasXaxpy(cublasv2handle,m,&alpha,dx+j*ldax,one,dy+j*lday,one);CHKERRCUBLAS(berr);
    }
  } else {
    berr = cublasXaxpy(cublasv2handle,N,&alpha,dx,one,dy,one);CHKERRCUBLAS(berr);
  }
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(PetscMax(2.*N-1,0));CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(X,&dx);CHKERRQ(ierr);
  if (alpha != 0.0) {
    ierr = MatDenseCUDARestoreArray(Y,&dy);CHKERRQ(ierr);
  } else {
    ierr = MatDenseCUDARestoreArrayWrite(Y,&dy);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatReset_SeqDenseCUDA(Mat A)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  hipError_t      cerr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (dA) {
    PetscCheckFalse(dA->unplacedarray,PETSC_COMM_SELF,PETSC_ERR_ORDER,"MatDenseCUDAResetArray() must be called first");
    if (!dA->user_alloc) { cerr = hipFree(dA->d_v);CHKERRCUDA(cerr); }
    cerr = hipFree(dA->d_fact_tau);CHKERRCUDA(cerr);
    cerr = hipFree(dA->d_fact_ipiv);CHKERRCUDA(cerr);
    cerr = hipFree(dA->d_fact_info);CHKERRCUDA(cerr);
    cerr = hipFree(dA->d_fact_work);CHKERRCUDA(cerr);
    ierr = VecDestroy(&dA->workvec);CHKERRQ(ierr);
  }
  ierr = PetscFree(A->spptr);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatDestroy_SeqDenseCUDA(Mat A)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  /* prevent to copy back data if we own the data pointer */
  if (!a->user_alloc) { A->offloadmask = PETSC_OFFLOAD_CPU; }
  ierr = MatConvert_SeqDenseCUDA_SeqDense(A,MATSEQDENSE,MAT_INPLACE_MATRIX,&A);CHKERRQ(ierr);
  ierr = MatDestroy_SeqDense(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatDuplicate_SeqDenseCUDA(Mat A,MatDuplicateOption cpvalues,Mat *B)
{
  PetscErrorCode     ierr;
  MatDuplicateOption hcpvalues = (cpvalues == MAT_COPY_VALUES && A->offloadmask != PETSC_OFFLOAD_CPU) ? MAT_DO_NOT_COPY_VALUES : cpvalues;

  PetscFunctionBegin;
  ierr = MatCreate(PetscObjectComm((PetscObject)A),B);CHKERRQ(ierr);
  ierr = MatSetSizes(*B,A->rmap->n,A->cmap->n,A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  ierr = MatSetType(*B,((PetscObject)A)->type_name);CHKERRQ(ierr);
  ierr = MatDuplicateNoCreate_SeqDense(*B,A,hcpvalues);CHKERRQ(ierr);
  if (cpvalues == MAT_COPY_VALUES && hcpvalues != MAT_COPY_VALUES) {
    ierr = MatCopy_SeqDenseCUDA(A,*B,SAME_NONZERO_PATTERN);CHKERRQ(ierr);
  }
  if (cpvalues != MAT_COPY_VALUES) { /* allocate memory if needed */
    Mat_SeqDenseCUDA *dB = (Mat_SeqDenseCUDA*)(*B)->spptr;
    if (!dB->d_v) {
      ierr = MatSeqDenseCUDASetPreallocation(*B,NULL);CHKERRQ(ierr);
    }
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatGetColumnVector_SeqDenseCUDA(Mat A,Vec v,PetscInt col)
{
  Mat_SeqDense     *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;
  PetscScalar      *x;
  PetscBool        viscuda;
  hipError_t      cerr;

  PetscFunctionBegin;
  ierr = PetscObjectTypeCompareAny((PetscObject)v,&viscuda,VECSEQCUDA,VECMPICUDA,VECCUDA,"");CHKERRQ(ierr);
  if (viscuda && !v->boundtocpu) { /* update device data */
    ierr = VecCUDAGetArrayWrite(v,&x);CHKERRQ(ierr);
    if (A->offloadmask & PETSC_OFFLOAD_GPU) {
      cerr = hipMemcpy(x,dA->d_v + col*a->lda,A->rmap->n*sizeof(PetscScalar),hipMemcpyHostToHost);CHKERRCUDA(cerr);
    } else {
      cerr = hipMemcpy(x,a->v + col*a->lda,A->rmap->n*sizeof(PetscScalar),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    }
    ierr = VecCUDARestoreArrayWrite(v,&x);CHKERRQ(ierr);
  } else { /* update host data */
    ierr = VecGetArrayWrite(v,&x);CHKERRQ(ierr);
    if (A->offloadmask == PETSC_OFFLOAD_UNALLOCATED || A->offloadmask & PETSC_OFFLOAD_CPU) {
      ierr = PetscArraycpy(x,a->v+col*a->lda,A->rmap->n);CHKERRQ(ierr);
    } else if (A->offloadmask & PETSC_OFFLOAD_GPU) {
      cerr = hipMemcpy(x,dA->d_v + col*a->lda,A->rmap->n*sizeof(PetscScalar),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    }
    ierr = VecRestoreArrayWrite(v,&x);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatGetFactor_seqdense_cuda(Mat A,MatFactorType ftype,Mat *fact)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate(PetscObjectComm((PetscObject)A),fact);CHKERRQ(ierr);
  ierr = MatSetSizes(*fact,A->rmap->n,A->cmap->n,A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  ierr = MatSetType(*fact,MATSEQDENSECUDA);CHKERRQ(ierr);
  if (ftype == MAT_FACTOR_LU || ftype == MAT_FACTOR_ILU) {
    (*fact)->ops->lufactorsymbolic = MatLUFactorSymbolic_SeqDense;
    (*fact)->ops->ilufactorsymbolic = MatLUFactorSymbolic_SeqDense;
  } else if (ftype == MAT_FACTOR_CHOLESKY || ftype == MAT_FACTOR_ICC) {
    (*fact)->ops->choleskyfactorsymbolic = MatCholeskyFactorSymbolic_SeqDense;
  } else if (ftype == MAT_FACTOR_QR) {
    ierr = PetscObjectComposeFunction((PetscObject)(*fact),"MatQRFactor_C",MatQRFactor_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)(*fact),"MatQRFactorSymbolic_C",MatQRFactorSymbolic_SeqDense);CHKERRQ(ierr);
  }
  (*fact)->factortype = ftype;
  ierr = PetscFree((*fact)->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATSOLVERCUDA,&(*fact)->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_LU]);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_ILU]);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_CHOLESKY]);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_ICC]);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetColumnVec_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckFalse(a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheckFalse(a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  ierr = MatDenseCUDAGetArray(A,(PetscScalar**)&a->ptrinuse);CHKERRQ(ierr);
  if (!a->cvec) { /* we pass the data of A, to prevent allocating needless GPU memory the first time VecCUDAPlaceArray is called */
    ierr = VecCreateSeqCUDAWithArray(PetscObjectComm((PetscObject)A),A->rmap->bs,A->rmap->n,a->ptrinuse,&a->cvec);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)A,(PetscObject)a->cvec);CHKERRQ(ierr);
  }
  a->vecinuse = col + 1;
  ierr = VecCUDAPlaceArray(a->cvec,a->ptrinuse + (size_t)col * (size_t)a->lda);CHKERRQ(ierr);
  *v   = a->cvec;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreColumnVec_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckFalse(!a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetColumnVec() first");
  PetscCheckFalse(!a->cvec,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column vector");
  a->vecinuse = 0;
  ierr = VecCUDAResetArray(a->cvec);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArray(A,(PetscScalar**)&a->ptrinuse);CHKERRQ(ierr);
  if (v) *v = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetColumnVecRead_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckFalse(a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheckFalse(a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  ierr = MatDenseCUDAGetArrayRead(A,&a->ptrinuse);CHKERRQ(ierr);
  if (!a->cvec) { /* we pass the data of A, to prevent allocating needless GPU memory the first time VecCUDAPlaceArray is called */
    ierr = VecCreateSeqCUDAWithArray(PetscObjectComm((PetscObject)A),A->rmap->bs,A->rmap->n,a->ptrinuse,&a->cvec);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)A,(PetscObject)a->cvec);CHKERRQ(ierr);
  }
  a->vecinuse = col + 1;
  ierr = VecCUDAPlaceArray(a->cvec,a->ptrinuse + (size_t)col * (size_t)a->lda);CHKERRQ(ierr);
  ierr = VecLockReadPush(a->cvec);CHKERRQ(ierr);
  *v   = a->cvec;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreColumnVecRead_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckFalse(!a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetColumnVec() first");
  PetscCheckFalse(!a->cvec,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column vector");
  a->vecinuse = 0;
  ierr = VecLockReadPop(a->cvec);CHKERRQ(ierr);
  ierr = VecCUDAResetArray(a->cvec);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&a->ptrinuse);CHKERRQ(ierr);
  if (v) *v = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetColumnVecWrite_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckFalse(a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheckFalse(a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  ierr = MatDenseCUDAGetArrayWrite(A,(PetscScalar**)&a->ptrinuse);CHKERRQ(ierr);
  if (!a->cvec) { /* we pass the data of A, to prevent allocating needless GPU memory the first time VecCUDAPlaceArray is called */
    ierr = VecCreateSeqCUDAWithArray(PetscObjectComm((PetscObject)A),A->rmap->bs,A->rmap->n,a->ptrinuse,&a->cvec);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)A,(PetscObject)a->cvec);CHKERRQ(ierr);
  }
  a->vecinuse = col + 1;
  ierr = VecCUDAPlaceArray(a->cvec,a->ptrinuse + (size_t)col * (size_t)a->lda);CHKERRQ(ierr);
  *v   = a->cvec;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreColumnVecWrite_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckFalse(!a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetColumnVec() first");
  PetscCheckFalse(!a->cvec,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column vector");
  a->vecinuse = 0;
  ierr = VecCUDAResetArray(a->cvec);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayWrite(A,(PetscScalar**)&a->ptrinuse);CHKERRQ(ierr);
  if (v) *v = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetSubMatrix_SeqDenseCUDA(Mat A,PetscInt cbegin,PetscInt cend,Mat *v)
{
  Mat_SeqDense     *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  PetscCheckFalse(a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheckFalse(a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  if (a->cmat && cend-cbegin != a->cmat->cmap->N) {
    ierr = MatDestroy(&a->cmat);CHKERRQ(ierr);
  }
  ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr);
  if (!a->cmat) {
    ierr = MatCreateDenseCUDA(PetscObjectComm((PetscObject)A),A->rmap->n,PETSC_DECIDE,A->rmap->N,cend-cbegin,dA->d_v+(size_t)cbegin*a->lda,&a->cmat);CHKERRQ(ierr);
    ierr = PetscLogObjectParent((PetscObject)A,(PetscObject)a->cmat);CHKERRQ(ierr);
  } else {
    ierr = MatDenseCUDAPlaceArray(a->cmat,dA->d_v+(size_t)cbegin*a->lda);CHKERRQ(ierr);
  }
  ierr = MatDenseSetLDA(a->cmat,a->lda);CHKERRQ(ierr);
  if (a->v) { ierr = MatDensePlaceArray(a->cmat,a->v+(size_t)cbegin*a->lda);CHKERRQ(ierr); }
  a->cmat->offloadmask = A->offloadmask;
  a->matinuse = cbegin + 1;
  *v = a->cmat;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreSubMatrix_SeqDenseCUDA(Mat A,Mat *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckFalse(!a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetSubMatrix() first");
  PetscCheckFalse(!a->cmat,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column matrix");
  PetscCheckFalse(*v != a->cmat,PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Not the matrix obtained from MatDenseGetSubMatrix()");
  a->matinuse = 0;
  A->offloadmask = (a->cmat->offloadmask == PETSC_OFFLOAD_CPU) ? PETSC_OFFLOAD_CPU : PETSC_OFFLOAD_GPU;
  ierr = MatDenseCUDAResetArray(a->cmat);CHKERRQ(ierr);
  if (a->unplacedarray) { ierr = MatDenseResetArray(a->cmat);CHKERRQ(ierr); }
  a->cmat->offloadmask = PETSC_OFFLOAD_UNALLOCATED;
  *v = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode  MatDenseSetLDA_SeqDenseCUDA(Mat A,PetscInt lda)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscBool        data;

  PetscFunctionBegin;
  data = (PetscBool)((A->rmap->n > 0 && A->cmap->n > 0) ? (dA->d_v ? PETSC_TRUE : PETSC_FALSE) : PETSC_FALSE);
  PetscCheckFalse(!dA->user_alloc && data && cA->lda!=lda,PETSC_COMM_SELF,PETSC_ERR_ORDER,"LDA cannot be changed after allocation of internal storage");
  PetscCheckFalse(lda < A->rmap->n,PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"LDA %" PetscInt_FMT " must be at least matrix dimension %" PetscInt_FMT,lda,A->rmap->n);
  cA->lda = lda;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSetUp_SeqDenseCUDA(Mat A)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = PetscLayoutSetUp(A->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->cmap);CHKERRQ(ierr);
  if (!A->preallocated) {
    ierr = MatSeqDenseCUDASetPreallocation(A,NULL);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatBindToCPU_SeqDenseCUDA(Mat A,PetscBool flg)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  PetscCheckFalse(a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheckFalse(a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  A->boundtocpu = flg;
  if (!flg) {
    PetscBool iscuda;

    ierr = PetscObjectTypeCompare((PetscObject)a->cvec,VECSEQCUDA,&iscuda);CHKERRQ(ierr);
    if (!iscuda) {
      ierr = VecDestroy(&a->cvec);CHKERRQ(ierr);
    }
    ierr = PetscObjectTypeCompare((PetscObject)a->cmat,MATSEQDENSECUDA,&iscuda);CHKERRQ(ierr);
    if (!iscuda) {
      ierr = MatDestroy(&a->cmat);CHKERRQ(ierr);
    }
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArray_C",MatDenseGetArray_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayRead_C",MatDenseGetArrayRead_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayWrite_C",MatDenseGetArrayWrite_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVec_C",MatDenseGetColumnVec_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVec_C",MatDenseRestoreColumnVec_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecRead_C",MatDenseGetColumnVecRead_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecRead_C",MatDenseRestoreColumnVecRead_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecWrite_C",MatDenseGetColumnVecWrite_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecWrite_C",MatDenseRestoreColumnVecWrite_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetSubMatrix_C",MatDenseGetSubMatrix_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreSubMatrix_C",MatDenseRestoreSubMatrix_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseSetLDA_C",MatDenseSetLDA_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatQRFactor_C",MatQRFactor_SeqDenseCUDA);CHKERRQ(ierr);

    A->ops->duplicate               = MatDuplicate_SeqDenseCUDA;
    A->ops->mult                    = MatMult_SeqDenseCUDA;
    A->ops->multadd                 = MatMultAdd_SeqDenseCUDA;
    A->ops->multtranspose           = MatMultTranspose_SeqDenseCUDA;
    A->ops->multtransposeadd        = MatMultTransposeAdd_SeqDenseCUDA;
    A->ops->matmultnumeric          = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->mattransposemultnumeric = MatMatTransposeMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->transposematmultnumeric = MatTransposeMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->axpy                    = MatAXPY_SeqDenseCUDA;
    A->ops->choleskyfactor          = MatCholeskyFactor_SeqDenseCUDA;
    A->ops->lufactor                = MatLUFactor_SeqDenseCUDA;
    A->ops->productsetfromoptions   = MatProductSetFromOptions_SeqDenseCUDA;
    A->ops->getcolumnvector         = MatGetColumnVector_SeqDenseCUDA;
    A->ops->scale                   = MatScale_SeqDenseCUDA;
    A->ops->copy                    = MatCopy_SeqDenseCUDA;
    A->ops->zeroentries             = MatZeroEntries_SeqDenseCUDA;
    A->ops->setup                   = MatSetUp_SeqDenseCUDA;
  } else {
    /* make sure we have an up-to-date copy on the CPU */
    ierr = MatSeqDenseCUDACopyFromGPU(A);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArray_C",MatDenseGetArray_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayRead_C",MatDenseGetArray_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayWrite_C",MatDenseGetArray_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVec_C",MatDenseGetColumnVec_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVec_C",MatDenseRestoreColumnVec_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecRead_C",MatDenseGetColumnVecRead_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecRead_C",MatDenseRestoreColumnVecRead_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecWrite_C",MatDenseGetColumnVecWrite_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecWrite_C",MatDenseRestoreColumnVecWrite_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetSubMatrix_C",MatDenseGetSubMatrix_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreSubMatrix_C",MatDenseRestoreSubMatrix_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseSetLDA_C",MatDenseSetLDA_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatQRFactor_C",MatQRFactor_SeqDense);CHKERRQ(ierr);

    A->ops->duplicate               = MatDuplicate_SeqDense;
    A->ops->mult                    = MatMult_SeqDense;
    A->ops->multadd                 = MatMultAdd_SeqDense;
    A->ops->multtranspose           = MatMultTranspose_SeqDense;
    A->ops->multtransposeadd        = MatMultTransposeAdd_SeqDense;
    A->ops->productsetfromoptions   = MatProductSetFromOptions_SeqDense;
    A->ops->matmultnumeric          = MatMatMultNumeric_SeqDense_SeqDense;
    A->ops->mattransposemultnumeric = MatMatTransposeMultNumeric_SeqDense_SeqDense;
    A->ops->transposematmultnumeric = MatTransposeMatMultNumeric_SeqDense_SeqDense;
    A->ops->axpy                    = MatAXPY_SeqDense;
    A->ops->choleskyfactor          = MatCholeskyFactor_SeqDense;
    A->ops->lufactor                = MatLUFactor_SeqDense;
    A->ops->productsetfromoptions   = MatProductSetFromOptions_SeqDense;
    A->ops->getcolumnvector         = MatGetColumnVector_SeqDense;
    A->ops->scale                   = MatScale_SeqDense;
    A->ops->copy                    = MatCopy_SeqDense;
    A->ops->zeroentries             = MatZeroEntries_SeqDense;
    A->ops->setup                   = MatSetUp_SeqDense;
  }
  if (a->cmat) {
    ierr = MatBindToCPU(a->cmat,flg);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatConvert_SeqDenseCUDA_SeqDense(Mat M,MatType type,MatReuse reuse,Mat *newmat)
{
  Mat              B;
  Mat_SeqDense     *a;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (reuse == MAT_REUSE_MATRIX || reuse == MAT_INITIAL_MATRIX) {
    /* TODO these cases should be optimized */
    ierr = MatConvert_Basic(M,type,reuse,newmat);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }

  B    = *newmat;
  ierr = MatBindToCPU_SeqDenseCUDA(B,PETSC_TRUE);CHKERRQ(ierr);
  ierr = MatReset_SeqDenseCUDA(B);CHKERRQ(ierr);
  ierr = PetscFree(B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(VECSTANDARD,&B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSEQDENSE);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatConvert_seqdensecuda_seqdense_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArray_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayRead_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayWrite_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArray_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayRead_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayWrite_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAPlaceArray_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAResetArray_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAReplaceArray_C",NULL);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatProductSetFromOptions_seqaij_seqdensecuda_C",NULL);CHKERRQ(ierr);
  a    = (Mat_SeqDense*)B->data;
  ierr = VecDestroy(&a->cvec);CHKERRQ(ierr); /* cvec might be VECSEQCUDA. Destroy it and rebuild a VECSEQ when needed */
  B->ops->bindtocpu = NULL;
  B->ops->destroy = MatDestroy_SeqDense;
  B->offloadmask = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

PetscErrorCode MatConvert_SeqDense_SeqDenseCUDA(Mat M,MatType type,MatReuse reuse,Mat *newmat)
{
  Mat_SeqDenseCUDA *dB;
  Mat              B;
  Mat_SeqDense     *a;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  ierr = PetscDeviceInitialize(PETSC_DEVICE_CUDA);CHKERRQ(ierr);
  if (reuse == MAT_REUSE_MATRIX || reuse == MAT_INITIAL_MATRIX) {
    /* TODO these cases should be optimized */
    ierr = MatConvert_Basic(M,type,reuse,newmat);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }

  B    = *newmat;
  ierr = PetscFree(B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(VECCUDA,&B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSEQDENSECUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatConvert_seqdensecuda_seqdense_C",            MatConvert_SeqDenseCUDA_SeqDense);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArray_C",                        MatDenseCUDAGetArray_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayRead_C",                    MatDenseCUDAGetArrayRead_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayWrite_C",                   MatDenseCUDAGetArrayWrite_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArray_C",                    MatDenseCUDARestoreArray_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayRead_C",                MatDenseCUDARestoreArrayRead_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayWrite_C",               MatDenseCUDARestoreArrayWrite_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAPlaceArray_C",                      MatDenseCUDAPlaceArray_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAResetArray_C",                      MatDenseCUDAResetArray_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAReplaceArray_C",                    MatDenseCUDAReplaceArray_SeqDenseCUDA);CHKERRQ(ierr);
  ierr = PetscObjectComposeFunction((PetscObject)B,"MatProductSetFromOptions_seqaij_seqdensecuda_C",MatProductSetFromOptions_SeqAIJ_SeqDense);CHKERRQ(ierr);
  a    = (Mat_SeqDense*)B->data;
  ierr = VecDestroy(&a->cvec);CHKERRQ(ierr); /* cvec might be VECSEQ. Destroy it and rebuild a VECSEQCUDA when needed */
  ierr = PetscNewLog(B,&dB);CHKERRQ(ierr);

  B->spptr = dB;
  B->offloadmask = PETSC_OFFLOAD_UNALLOCATED;

  ierr = MatBindToCPU_SeqDenseCUDA(B,PETSC_FALSE);CHKERRQ(ierr);
  B->ops->bindtocpu = MatBindToCPU_SeqDenseCUDA;
  B->ops->destroy  = MatDestroy_SeqDenseCUDA;
  PetscFunctionReturn(0);
}

/*@C
   MatCreateSeqDenseCUDA - Creates a sequential matrix in dense format using CUDA.

   Collective

   Input Parameters:
+  comm - MPI communicator
.  m - number of rows
.  n - number of columns
-  data - optional location of GPU matrix data.  Set data=NULL for PETSc
   to control matrix memory allocation.

   Output Parameter:
.  A - the matrix

   Notes:

   Level: intermediate

.seealso: MatCreate(), MatCreateSeqDense()
@*/
PetscErrorCode  MatCreateSeqDenseCUDA(MPI_Comm comm,PetscInt m,PetscInt n,PetscScalar *data,Mat *A)
{
  PetscErrorCode ierr;
  PetscMPIInt    size;

  PetscFunctionBegin;
  ierr = MPI_Comm_size(comm,&size);CHKERRMPI(ierr);
  PetscCheckFalse(size > 1,comm,PETSC_ERR_ARG_WRONG,"Invalid communicator size %d",size);
  ierr = MatCreate(comm,A);CHKERRQ(ierr);
  ierr = MatSetSizes(*A,m,n,m,n);CHKERRQ(ierr);
  ierr = MatSetType(*A,MATSEQDENSECUDA);CHKERRQ(ierr);
  ierr = MatSeqDenseCUDASetPreallocation(*A,data);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*MC
   MATSEQDENSECUDA - MATSEQDENSECUDA = "seqdensecuda" - A matrix type to be used for sequential dense matrices on GPUs.

   Options Database Keys:
. -mat_type seqdensecuda - sets the matrix type to "seqdensecuda" during a call to MatSetFromOptions()

  Level: beginner
M*/
PETSC_EXTERN PetscErrorCode MatCreate_SeqDenseCUDA(Mat B)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = PetscDeviceInitialize(PETSC_DEVICE_CUDA);CHKERRQ(ierr);
  ierr = MatCreate_SeqDense(B);CHKERRQ(ierr);
  ierr = MatConvert_SeqDense_SeqDenseCUDA(B,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
