#include "hip/hip_runtime.h"
/*
   Implements the Landau kernel
*/
#include <petscconf.h>
#include <petsc/private/dmpleximpl.h>   /*I  "dmpleximpl.h"   I*/
#include <petsclandau.h>
#define PETSC_SKIP_IMMINTRIN_H_CUDAWORKAROUND 1
#include <../src/mat/impls/aij/seq/aij.h>
#include <petscmat.h>
#include <petsccublas.h>

// hack to avoid configure problems in CI. Delete when resolved
#if !defined (PETSC_HAVE_CUDA_ATOMIC)
#define atomicAdd(e, f) (*e) += f
#endif
#define PETSC_DEVICE_FUNC_DECL __device__
#include "../land_tensors.h"
#include <petscaijdevice.h>

// Macro to catch CUDA errors in CUDA runtime calls
#define CUDA_SAFE_CALL(call)                                          \
do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)
// Macro to catch CUDA errors in kernel launches
#define CHECK_LAUNCH_ERROR()                                          \
do {                                                                  \
    /* Check synchronous errors, i.e. pre-launch */                   \
    hipError_t err = hipGetLastError();                             \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
    /* Check asynchronous errors, i.e. kernel failed (ULF) */         \
    err = hipDeviceSynchronize();                                    \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString( err) );      \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

PETSC_EXTERN PetscErrorCode LandauCUDACreateMatMaps(P4estVertexMaps *maps, pointInterpolationP4est (*points)[LANDAU_MAX_Q_FACE], PetscInt Nf, PetscInt Nq)
{
  P4estVertexMaps h_maps;
  PetscFunctionBegin;
  h_maps.num_elements =maps->num_elements;
  h_maps.num_face = maps->num_face;
  h_maps.num_reduced = maps->num_reduced;
  h_maps.deviceType = maps->deviceType;
  h_maps.Nf = Nf;
  h_maps.Nq = Nq;
  CUDA_SAFE_CALL(hipMalloc((void **)&h_maps.c_maps,               maps->num_reduced  * sizeof *points));
  CUDA_SAFE_CALL(hipMemcpy(          h_maps.c_maps, maps->c_maps, maps->num_reduced  * sizeof *points, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&h_maps.gIdx,                 maps->num_elements * sizeof *maps->gIdx));
  CUDA_SAFE_CALL(hipMemcpy(          h_maps.gIdx, maps->gIdx,     maps->num_elements * sizeof *maps->gIdx, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&maps->data, sizeof(P4estVertexMaps)));
  CUDA_SAFE_CALL(hipMemcpy(          maps->data,   &h_maps, sizeof(P4estVertexMaps), hipMemcpyHostToDevice));
  PetscFunctionReturn(0);
}

PETSC_EXTERN PetscErrorCode LandauCUDADestroyMatMaps(P4estVertexMaps *pMaps)
{
  P4estVertexMaps *d_maps = pMaps->data, h_maps;
  PetscFunctionBegin;
  CUDA_SAFE_CALL(hipMemcpy(&h_maps, d_maps, sizeof(P4estVertexMaps), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(h_maps.c_maps));
  CUDA_SAFE_CALL(hipFree(h_maps.gIdx));
  CUDA_SAFE_CALL(hipFree(d_maps));
  PetscFunctionReturn(0);
}

// The GPU Landau kernel
//
__global__
void __launch_bounds__(256,1) landau_form_fdf(const PetscInt nip, const PetscInt dim, const PetscInt Nf, const PetscInt Nb, const PetscReal invJ_a[],
					      const PetscReal * const BB, const PetscReal * const DD, LandauIPReal *IPDataRaw, LandauIPReal d_f[], LandauIPReal d_dfdx[], LandauIPReal d_dfdy[], 
#if LANDAU_DIM==3
					      LandauIPReal d_dfdz[],
#endif
					      PetscErrorCode *ierr) // output
{
  const PetscInt  Nq = blockDim.y, myelem = blockIdx.x;
  const PetscInt  myQi = threadIdx.y;
  const PetscInt  jpidx = myQi + myelem * Nq;
  const PetscReal *invJ = &invJ_a[jpidx*dim*dim];
  const PetscReal *Bq = &BB[myQi*Nb], *Dq = &DD[myQi*Nb*dim];
  // un pack IPData
  LandauIPReal    *IPData_coefs = &IPDataRaw[nip*(dim+1)];
  LandauIPReal    *coef = &IPData_coefs[myelem*Nb*Nf];
  PetscInt        f,d,b,e;
  PetscScalar     u_x[LANDAU_MAX_SPECIES][LANDAU_DIM];
  *ierr = 0;
  /* get f and df */
  for (f = 0; f < Nf; ++f) {
    PetscScalar refSpaceDer[LANDAU_DIM];
    d_f[jpidx + f*nip] = 0.0;
    for (d = 0; d < LANDAU_DIM; ++d) refSpaceDer[d] = 0.0;
    for (b = 0; b < Nb; ++b) {
      const PetscInt    cidx = b;
      d_f[jpidx + f*nip] += Bq[cidx]*coef[f*Nb+cidx];
      for (d = 0; d < dim; ++d) refSpaceDer[d] += Dq[cidx*dim+d]*coef[f*Nb+cidx];
    }
    for (d = 0; d < dim; ++d) {
      for (e = 0, u_x[f][d] = 0.0; e < dim; ++e) {
	u_x[f][d] += invJ[e*dim+d]*refSpaceDer[e];
      }
    }
  }
  for (f=0;f<Nf;f++) {
    d_dfdx[jpidx + f*nip] = PetscRealPart(u_x[f][0]);
    d_dfdy[jpidx + f*nip] = PetscRealPart(u_x[f][1]);
#if LANDAU_DIM==3
    d_dfdz[jpidx + f*nip] = PetscRealPart(u_x[f][2]);
#endif
  }
}

__device__ void
landau_inner_integral_v2(const PetscInt myQi, const PetscInt jpidx, PetscInt nip, const PetscInt Nq, const PetscInt Nf, const PetscInt Nb,
			 const PetscInt dim, LandauIPReal *IPDataRaw, const PetscReal invJj[], const PetscReal nu_alpha[],
			 const PetscReal nu_beta[], const PetscReal invMass[], const PetscReal Eq_m[],
			 const PetscReal * const BB, const PetscReal * const DD,
			 PetscScalar *elemMat, P4estVertexMaps *d_maps, PetscSplitCSRDataStructure *d_mat, // output
			 PetscScalar fieldMats[][LANDAU_MAX_NQ], // all these arrays are in shared memory
			 PetscReal g2[][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES],
			 PetscReal g3[][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES],
			 PetscReal gg2[][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES],
			 PetscReal gg3[][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES],
			 PetscReal s_nu_alpha[],
			 PetscReal s_nu_beta[],
			 PetscReal s_invMass[],
			 PetscReal s_f[],
			 PetscReal s_dfx[],
			 PetscReal s_dfy[],
			 LandauIPReal d_f[], LandauIPReal d_dfdx[], LandauIPReal d_dfdy[], // global memory
#if LANDAU_DIM==3
			 PetscReal s_dfz[], LandauIPReal d_dfdz[],
#endif
			 PetscInt myelem, PetscErrorCode *ierr)
{
  int           delta,d,f,g,d2,dp,d3,fieldA,ipidx_b,nip_pad = nip; // vectorization padding not supported;
  PetscReal     gg2_temp[LANDAU_DIM], gg3_temp[LANDAU_DIM][LANDAU_DIM];
  LandauIPData  IPData;

  *ierr = 0;
  // create g2 & g3
  for (f=threadIdx.x; f<Nf; f+=blockDim.x) {
    for (d=0;d<dim;d++) { // clear accumulation data D & K
      gg2[d][myQi][f] = 0;
      for (d2=0;d2<dim;d2++) gg3[d][d2][myQi][f] = 0;
    }
  }
  if (threadIdx.y == 0) {
    for (int i = threadIdx.x; i < Nf; i += blockDim.x) {
      s_nu_alpha[i] = nu_alpha[i];
      s_nu_beta[i] = nu_beta[i];
      s_invMass[i] = invMass[i];
    }
  }
  for (d2 = 0; d2 < dim; d2++) {
    gg2_temp[d2] = 0;
    for (d3 = 0; d3 < dim; d3++) {
      gg3_temp[d2][d3] = 0;
    }
  }
  __syncthreads();
  // un pack IPData
  IPData.w   = IPDataRaw;
  IPData.x   = IPDataRaw + 1*nip_pad;
  IPData.y   = IPDataRaw + 2*nip_pad;
  IPData.z   = IPDataRaw + 3*nip_pad;

  const PetscReal vj[3] = {IPData.x[jpidx], IPData.y[jpidx], IPData.z ? IPData.z[jpidx] : 0}, wj = IPData.w[jpidx];
  for (ipidx_b = 0; ipidx_b < nip; ipidx_b += blockDim.x) {
    int ipidx = ipidx_b + threadIdx.x;

    __syncthreads();
    if (ipidx < nip) {
      for (fieldA = threadIdx.y; fieldA < Nf; fieldA += blockDim.y) {
        s_f  [fieldA*blockDim.x+threadIdx.x] =    d_f[ipidx + fieldA*nip_pad];
        s_dfx[fieldA*blockDim.x+threadIdx.x] = d_dfdx[ipidx + fieldA*nip_pad];
        s_dfy[fieldA*blockDim.x+threadIdx.x] = d_dfdy[ipidx + fieldA*nip_pad];
#if LANDAU_DIM==3
        s_dfz[fieldA*blockDim.x+threadIdx.x] = d_dfdz[ipidx + fieldA*nip_pad];
#endif
      }
    }
    __syncthreads();
    if (ipidx < nip) {
      const PetscReal wi = IPData.w[ipidx], x = IPData.x[ipidx], y = IPData.y[ipidx];
      PetscReal       temp1[3] = {0, 0, 0}, temp2 = 0;
#if LANDAU_DIM==2
      PetscReal Ud[2][2], Uk[2][2];
      LandauTensor2D(vj, x, y, Ud, Uk, (ipidx==jpidx) ? 0. : 1.);
#else
      PetscReal U[3][3], z = IPData.z[ipidx];
      LandauTensor3D(vj, x, y, z, U, (ipidx==jpidx) ? 0. : 1.);
#endif
      for (fieldA = 0; fieldA < Nf; fieldA++) {
        temp1[0] += s_dfx[fieldA*blockDim.x+threadIdx.x]*s_nu_beta[fieldA]*s_invMass[fieldA];
        temp1[1] += s_dfy[fieldA*blockDim.x+threadIdx.x]*s_nu_beta[fieldA]*s_invMass[fieldA];
#if LANDAU_DIM==3
        temp1[2] += s_dfz[fieldA*blockDim.x+threadIdx.x]*s_nu_beta[fieldA]*s_invMass[fieldA];
#endif
        temp2    += s_f  [fieldA*blockDim.x+threadIdx.x]*s_nu_beta[fieldA];
      }
      temp1[0] *= wi;
      temp1[1] *= wi;
#if LANDAU_DIM==3
      temp1[2] *= wi;
#endif
      temp2    *= wi;
#if LANDAU_DIM==2
      for (d2 = 0; d2 < 2; d2++) {
        for (d3 = 0; d3 < 2; ++d3) {
          /* K = U * grad(f): g2=e: i,A */
          gg2_temp[d2] += Uk[d2][d3]*temp1[d3];
          /* D = -U * (I \kron (fx)): g3=f: i,j,A */
          gg3_temp[d2][d3] += Ud[d2][d3]*temp2;
        }
      }
      #else
      for (d2 = 0; d2 < 3; ++d2) {
        for (d3 = 0; d3 < 3; ++d3) {
          /* K = U * grad(f): g2 = e: i,A */
          gg2_temp[d2] += U[d2][d3]*temp1[d3];
          /* D = -U * (I \kron (fx)): g3 = f: i,j,A */
          gg3_temp[d2][d3] += U[d2][d3]*temp2;
        }
      }
      #endif
    }
  } /* IPs */
 

  /* reduce gg temp sums across threads */
  for (delta = blockDim.x/2; delta > 0; delta /= 2) {
    for (d2 = 0; d2 < dim; d2++) {
      gg2_temp[d2] += __shfl_xor_sync(0xffffffff, gg2_temp[d2], delta, blockDim.x);
      for (d3 = 0; d3 < dim; d3++) {
        gg3_temp[d2][d3] += __shfl_xor_sync(0xffffffff, gg3_temp[d2][d3], delta, blockDim.x);
      }
    }
  }

  // add alpha and put in gg2/3
  for (fieldA = threadIdx.x; fieldA < Nf; fieldA += blockDim.x) {
    for (d2 = 0; d2 < dim; d2++) {
      gg2[d2][myQi][fieldA] += gg2_temp[d2]*s_nu_alpha[fieldA];
      for (d3 = 0; d3 < dim; d3++) {
        gg3[d2][d3][myQi][fieldA] -= gg3_temp[d2][d3]*s_nu_alpha[fieldA]*s_invMass[fieldA];
      }
    }
  }
  __syncthreads();

  /* add electric field term once per IP */
  for (fieldA = threadIdx.x; fieldA < Nf; fieldA += blockDim.x) {
    gg2[dim-1][myQi][fieldA] += Eq_m[fieldA];
  }
  __syncthreads();
  //intf("%d %d gg2[1][1]=%g\n",myelem,qj_start,gg2[1][dim-1]);
  /* Jacobian transform - g2 */
  for (fieldA = threadIdx.x; fieldA < Nf; fieldA += blockDim.x) {
    for (d = 0; d < dim; ++d) {
      g2[d][myQi][fieldA] = 0.0;
      for (d2 = 0; d2 < dim; ++d2) {
        g2[d][myQi][fieldA] += invJj[d*dim+d2]*gg2[d2][myQi][fieldA];
        g3[d][d2][myQi][fieldA] = 0.0;
        for (d3 = 0; d3 < dim; ++d3) {
          for (dp = 0; dp < dim; ++dp) {
            g3[d][d2][myQi][fieldA] += invJj[d*dim + d3]*gg3[d3][dp][myQi][fieldA]*invJj[d2*dim + dp];
          }
        }
        g3[d][d2][myQi][fieldA] *= wj;
      }
      g2[d][myQi][fieldA] *= wj;
    }
  }
  /* FE matrix construction */
  __syncthreads();  // Synchronize (ensure all the data is available) and sum IP matrices
  {
    int fieldA,d,qj,d2,q,idx,totDim=Nb*Nf;
    /* assemble */
    for (fieldA = 0; fieldA < Nf; fieldA++) {
      if (fieldMats) {
	for (f = threadIdx.y; f < Nb ; f += blockDim.y) {
	  for (g = threadIdx.x; g < Nb; g += blockDim.x) {
	    fieldMats[f][g] = 0;
	  }
	}
      }
      for (f = threadIdx.y; f < Nb ; f += blockDim.y) {
        const PetscInt i = fieldA*Nb + f; /* Element matrix row */
        for (g = threadIdx.x; g < Nb; g += blockDim.x) {
          const PetscInt j    = fieldA*Nb + g; /* Element matrix column */
          const PetscInt fOff = i*totDim + j;
          PetscScalar t = elemMat ? elemMat[fOff] : fieldMats[f][g];
          for (qj = 0 ; qj < Nq ; qj++) {
            const PetscReal *BJq = &BB[qj*Nb], *DIq = &DD[qj*Nb*dim];
            for (d = 0; d < dim; ++d) {
              t += DIq[f*dim+d]*g2[d][qj][fieldA]*BJq[g];
              for (d2 = 0; d2 < dim; ++d2) {
                t += DIq[f*dim + d]*g3[d][d2][qj][fieldA]*DIq[g*dim + d2];
              }
            }
          }
	  if (elemMat) elemMat[fOff] = t;
	  else fieldMats[f][g] = t;
	}
      }
      if (fieldMats) {
	PetscScalar            vals[LANDAU_MAX_Q*LANDAU_MAX_Q];
	PetscReal              row_scale[LANDAU_MAX_Q],col_scale[LANDAU_MAX_Q];
	PetscInt               nr,nc,rows0[LANDAU_MAX_Q],cols0[LANDAU_MAX_Q],rows[LANDAU_MAX_Q],cols[LANDAU_MAX_Q];
	const LandauIdx *const Idxs = &d_maps->gIdx[myelem][fieldA][0];
	for (f = threadIdx.y; f < Nb ; f += blockDim.y) {
	  idx = Idxs[f];
	  if (idx >= 0) {
	    nr = 1;
	    rows0[0] = idx;
	    row_scale[0] = 1.;
	  } else {
	    idx = -idx - 1;
	    nr = d_maps->num_face;
	    for (q = 0; q < d_maps->num_face; q++) {
	      rows0[q]     = d_maps->c_maps[idx][q].gid;
	      row_scale[q] = d_maps->c_maps[idx][q].scale;
	    }
	  }
	  for (g = threadIdx.x; g < Nb; g += blockDim.x) {
	    idx = Idxs[g];
	    if (idx >= 0) {
	      nc = 1;
	      cols0[0] = idx;
	      col_scale[0] = 1.;
	    } else {
	      idx = -idx - 1;
	      nc = d_maps->num_face;
	      for (q = 0; q < d_maps->num_face; q++) {
		cols0[q]     = d_maps->c_maps[idx][q].gid;
		col_scale[q] = d_maps->c_maps[idx][q].scale;
	      }
	    }
	    for (q = 0; q < nr; q++) rows[q] = rows0[q];
	    for (q = 0; q < nc; q++) cols[q] = cols0[q];
	    for (q = 0; q < nr; q++) {
	      for (d = 0; d < nc; d++) {
		vals[q*nc + d] = row_scale[q]*col_scale[d]*fieldMats[f][g];
	      }
	    }
	    MatSetValuesDevice(d_mat,nr,rows,nc,cols,vals,ADD_VALUES,ierr);
	    if (*ierr) return;
	  }
	}
      }
    }
  }
}

//
// The GPU Landau kernel
//
__global__
void __launch_bounds__(256,1) landau_kernel_v2(const PetscInt nip, const PetscInt dim, const PetscInt totDim, const PetscInt Nf, const PetscInt Nb, const PetscReal invJj[],
  const PetscReal nu_alpha[], const PetscReal nu_beta[], const PetscReal invMass[], const PetscReal Eq_m[],
  const PetscReal * const BB, const PetscReal * const DD, LandauIPReal *IPDataRaw,
  PetscScalar elemMats_out[], P4estVertexMaps *d_maps, PetscSplitCSRDataStructure *d_mat, LandauIPReal d_f[], LandauIPReal d_dfdx[], LandauIPReal d_dfdy[], 
#if LANDAU_DIM==3
					       LandauIPReal d_dfdz[],
#endif
					       PetscErrorCode *ierr)
{
  const PetscInt  Nq = blockDim.y, myelem = blockIdx.x;
  extern __shared__ PetscReal smem[];
  int size = 0;
  PetscReal (*g2)[LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES]              =
    (PetscReal (*)[LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES])             &smem[size];
  size += LANDAU_MAX_NQ*LANDAU_MAX_SPECIES*LANDAU_DIM;
  PetscReal (*g3)[LANDAU_DIM][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES]  =
    (PetscReal (*)[LANDAU_DIM][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES]) &smem[size];
  size += LANDAU_DIM*LANDAU_DIM*LANDAU_MAX_NQ*LANDAU_MAX_SPECIES;
  PetscReal (*gg2)[LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES]             =
    (PetscReal (*)[LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES])             &smem[size];
  size += LANDAU_MAX_NQ*LANDAU_MAX_SPECIES*LANDAU_DIM;
  PetscReal (*gg3)[LANDAU_DIM][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES] =
    (PetscReal (*)[LANDAU_DIM][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES]) &smem[size];
  size += LANDAU_DIM*LANDAU_DIM*LANDAU_MAX_NQ*LANDAU_MAX_SPECIES;
  PetscReal *s_nu_alpha = &smem[size];
  size += LANDAU_MAX_SPECIES;
  PetscReal *s_nu_beta  = &smem[size];
  size += LANDAU_MAX_SPECIES;
  PetscReal *s_invMass  = &smem[size];
  size += LANDAU_MAX_SPECIES;
  PetscReal *s_f        = &smem[size];
  size += blockDim.x*LANDAU_MAX_SPECIES;
  PetscReal *s_dfx      = &smem[size];
  size += blockDim.x*LANDAU_MAX_SPECIES;
  PetscReal *s_dfy      = &smem[size];
  size += blockDim.x*LANDAU_MAX_SPECIES;
#if LANDAU_DIM==3
  PetscReal *s_dfz      = &smem[size];
  size += blockDim.x*LANDAU_MAX_SPECIES;
#endif
  PetscScalar (*fieldMats)[LANDAU_MAX_NQ][LANDAU_MAX_NQ] = d_maps ?
    (PetscScalar (*)[LANDAU_MAX_NQ][LANDAU_MAX_NQ]) &smem[size] : NULL;
  if (d_maps) size += LANDAU_MAX_NQ*LANDAU_MAX_NQ;
  const PetscInt  myQi = threadIdx.y;
  const PetscInt  jpidx = myQi + myelem * Nq;
  //const PetscInt  subblocksz = nip/nSubBlks + !!(nip%nSubBlks), ip_start = mySubBlk*subblocksz, ip_end = (mySubBlk+1)*subblocksz > nip ? nip : (mySubBlk+1)*subblocksz; /* this could be wrong with very few global IPs */
  PetscScalar     *elemMat  = elemMats_out ? &elemMats_out[myelem*totDim*totDim] : NULL; /* my output */
  int tid = threadIdx.x + threadIdx.y*blockDim.x;

  if (elemMat) for (int i = tid; i < totDim*totDim; i += blockDim.x*blockDim.y) elemMat[i] = 0;
  __syncthreads();

  landau_inner_integral_v2(myQi, jpidx, nip, Nq, Nf, Nb, dim, IPDataRaw, &invJj[jpidx*dim*dim], nu_alpha, nu_beta, invMass, Eq_m, BB, DD,
			   elemMat, d_maps, d_mat, *fieldMats, *g2, *g3, *gg2, *gg3, s_nu_alpha, s_nu_beta, s_invMass, s_f, s_dfx, s_dfy, d_f, d_dfdx, d_dfdy,
#if LANDAU_DIM==3
			   s_dfz, d_dfdz,
#endif
			   myelem, ierr); /* compact */
}

PetscErrorCode LandauCUDAJacobian(DM plex, const PetscInt Nq, const PetscReal nu_alpha[],const PetscReal nu_beta[], const PetscReal invMass[], const PetscReal Eq_m[],
				  const LandauIPData *const IPData, const PetscReal invJj[], const PetscLogEvent events[], Mat JacP)
{
  PetscErrorCode    ierr,*d_ierr;
  //hipError_t       cerr;
  PetscInt          ii,ej,*Nbf,Nb,nip_dim2,cStart,cEnd,Nf,dim,numGCells,totDim,nip,szf=sizeof(LandauIPReal),ipdatasz;
  PetscReal         *d_BB,*d_DD,*d_invJj,*d_nu_alpha,*d_nu_beta,*d_invMass,*d_Eq_m;
  PetscScalar       *d_elemMats=NULL;
  LandauIPReal       *d_f, *d_dfdx, *d_dfdy;
#if LANDAU_DIM==3
  PetscScalar       *d_dfdz;
#endif
  PetscLogDouble    flops;
  PetscTabulation   *Tf;
  PetscDS           prob;
  PetscSection      section, globalSection;
  LandauIPReal      *d_IPDataRaw;
  LandauCtx         *ctx;
  PetscSplitCSRDataStructure *d_mat=NULL;
  P4estVertexMaps            *h_maps, *d_maps=NULL;
  int               nnn = 256/Nq;

  PetscFunctionBegin;
  while (nnn & nnn - 1) nnn = nnn & nnn - 1;
  if (nnn>16) nnn = 16;
  ierr = PetscLogEventBegin(events[3],0,0,0,0);CHKERRQ(ierr);
  ierr = DMGetDimension(plex, &dim);CHKERRQ(ierr);
  if (dim!=LANDAU_DIM) SETERRQ2(PETSC_COMM_SELF, PETSC_ERR_PLIB, "LANDAU_DIM %D != dim %d",LANDAU_DIM,dim);
  ierr = DMPlexGetHeightStratum(plex,0,&cStart,&cEnd);CHKERRQ(ierr);
  numGCells = cEnd - cStart;
  nip  = numGCells*Nq; /* length of inner global iteration */
  ierr = DMGetDS(plex, &prob);CHKERRQ(ierr);
  ierr = PetscDSGetNumFields(prob, &Nf);CHKERRQ(ierr);
  ierr = PetscDSGetDimensions(prob, &Nbf);CHKERRQ(ierr); Nb = Nbf[0];
  if (Nq != Nb) SETERRQ2(PETSC_COMM_SELF, PETSC_ERR_PLIB, "Nq != Nb. %D  %D",Nq,Nb);
  ierr = PetscDSGetTotalDimension(prob, &totDim);CHKERRQ(ierr);
  ierr = PetscDSGetTabulation(prob, &Tf);CHKERRQ(ierr);
  ierr = DMGetLocalSection(plex, &section);CHKERRQ(ierr);
  ierr = DMGetGlobalSection(plex, &globalSection);CHKERRQ(ierr);
  ipdatasz = LandauGetIPDataSize(IPData);
  // create data
  CUDA_SAFE_CALL(hipMalloc((void **)&d_IPDataRaw,ipdatasz*szf )); // kernel input
  CUDA_SAFE_CALL(hipMalloc((void **)&d_nu_alpha, Nf*szf)); // kernel input
  CUDA_SAFE_CALL(hipMalloc((void **)&d_nu_beta,  Nf*szf)); // kernel input
  CUDA_SAFE_CALL(hipMalloc((void **)&d_invMass,  Nf*szf)); // kernel input
  CUDA_SAFE_CALL(hipMalloc((void **)&d_Eq_m,     Nf*szf)); // kernel input
  CUDA_SAFE_CALL(hipMemcpy(d_IPDataRaw, IPData->w, ipdatasz*szf, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_nu_alpha, nu_alpha, Nf*szf,       hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_nu_beta,  nu_beta,  Nf*szf,       hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_invMass,  invMass,  Nf*szf,       hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_Eq_m,     Eq_m,     Nf*szf,       hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_BB,              Nq*Nb*szf));     // kernel input
  CUDA_SAFE_CALL(hipMemcpy(          d_BB, Tf[0]->T[0], Nq*Nb*szf,   hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_DD,              Nq*Nb*dim*szf)); // kernel input
  CUDA_SAFE_CALL(hipMemcpy(          d_DD, Tf[0]->T[1], Nq*Nb*dim*szf,   hipMemcpyHostToDevice));
  // f and df
  CUDA_SAFE_CALL(hipMalloc((void **)&d_f,    nip*Nf*szf));     // kernel input
  CUDA_SAFE_CALL(hipMalloc((void **)&d_dfdx, nip*Nf*szf));     // kernel input
  CUDA_SAFE_CALL(hipMalloc((void **)&d_dfdy, nip*Nf*szf));     // kernel input
#if LANDAU_DIM==3
  CUDA_SAFE_CALL(hipMalloc((void **)&d_dfdz, nip*Nf*szf));     // kernel input
#endif
  // collect geometry
  flops = (PetscLogDouble)numGCells*(PetscLogDouble)Nq*(PetscLogDouble)(5.*dim*dim*Nf*Nf + 165.);
  nip_dim2 = Nq*numGCells*dim*dim;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_invJj, nip_dim2*szf)); // kernel input
  CUDA_SAFE_CALL(hipMemcpy(d_invJj, invJj, nip_dim2*szf,       hipMemcpyHostToDevice));
  //cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogEventEnd(events[3],0,0,0,0);CHKERRQ(ierr);

  ierr = PetscLogEventBegin(events[4],0,0,0,0);CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(flops*nip);CHKERRQ(ierr);
  ierr = DMGetApplicationContext(plex, &ctx);CHKERRQ(ierr);
  if (!ctx) SETERRQ(PETSC_COMM_SELF, PETSC_ERR_PLIB, "no context");
  if (ctx->gpu_assembly) {
    PetscContainer container;
    ierr = PetscObjectQuery((PetscObject) JacP, "assembly_maps", (PetscObject *) &container);CHKERRQ(ierr);
    if (container) { // not here first call
      ierr = PetscContainerGetPointer(container, (void **) &h_maps);CHKERRQ(ierr);
      if (h_maps->data) {
        d_maps = h_maps->data;
	if (!d_maps) SETERRQ(PETSC_COMM_SELF, PETSC_ERR_PLIB, "GPU assembly but no metadata");
      } else {
        SETERRQ(PETSC_COMM_SELF, PETSC_ERR_PLIB, "GPU assembly but no metadata in container");
      }
      // this does the setup the first time called
      ierr = MatCUSPARSEGetDeviceMatWrite(JacP,&d_mat);CHKERRQ(ierr);
    } else {
      CUDA_SAFE_CALL(hipMalloc((void **)&d_elemMats, totDim*totDim*numGCells*sizeof(PetscScalar))); // kernel output - first call is on CPU
    }
  } else {
    CUDA_SAFE_CALL(hipMalloc((void **)&d_elemMats, totDim*totDim*numGCells*sizeof(PetscScalar))); // kernel output - no GPU assembly
  }
  CUDA_SAFE_CALL(hipMalloc((void **)&d_ierr, sizeof(ierr))); // kernel input
  { // form f and df
    dim3 dimBlock(nnn,Nq);
    ierr = PetscLogEventBegin(events[8],0,0,0,0);CHKERRQ(ierr);
    ii = 0;
    // PetscPrintf(PETSC_COMM_SELF, "numGCells=%d dim.x=%d Nq=%d nThreads=%d, %d kB shared mem\n",numGCells,n,Nq,Nq*n,ii*szf/1024);
    landau_form_fdf<<<numGCells,dimBlock,ii*szf>>>( nip, dim, Nf, Nb, d_invJj, d_BB, d_DD, d_IPDataRaw, d_f, d_dfdx, d_dfdy,
#if LANDAU_DIM==3
						    d_dfdz,
#endif
						    d_ierr);
    CHECK_LAUNCH_ERROR();
    CUDA_SAFE_CALL(hipMemcpy(&ierr, d_ierr, sizeof(ierr), hipMemcpyDeviceToHost));
    CHKERRQ(ierr);
    ierr = PetscLogEventEnd(events[8],0,0,0,0);CHKERRQ(ierr);
  }
  {
    dim3 dimBlock(nnn,Nq);
    ii = 2*LANDAU_MAX_NQ*LANDAU_MAX_SPECIES*LANDAU_DIM*(1+LANDAU_DIM) + 3*LANDAU_MAX_SPECIES + (1+LANDAU_DIM)*dimBlock.x*LANDAU_MAX_SPECIES;
    ii += (LANDAU_MAX_NQ*LANDAU_MAX_NQ)*LANDAU_MAX_SPECIES;
    if (ii*szf >= 49152) {
      CUDA_SAFE_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(landau_kernel_v2),
					  hipFuncAttributeMaxDynamicSharedMemorySize,
					  98304));
    }
    // PetscPrintf(PETSC_COMM_SELF, "numGCells=%d dim.x=%d Nq=%d nThreads=%d, %d kB shared mem\n",numGCells,n,Nq,Nq*n,ii*szf/1024);
    landau_kernel_v2<<<numGCells,dimBlock,ii*szf>>>(nip,dim,totDim,Nf,Nb,d_invJj,d_nu_alpha,d_nu_beta,d_invMass,d_Eq_m,
						    d_BB, d_DD, d_IPDataRaw, d_elemMats, d_maps, d_mat, d_f, d_dfdx, d_dfdy,
#if LANDAU_DIM==3
						    d_dfdz,
#endif
						    d_ierr);
    CHECK_LAUNCH_ERROR();
    CUDA_SAFE_CALL(hipMemcpy(&ierr, d_ierr, sizeof(ierr), hipMemcpyDeviceToHost));
    CHKERRQ(ierr);
  }
  CUDA_SAFE_CALL(hipFree(d_ierr));
  //cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogEventEnd(events[4],0,0,0,0);CHKERRQ(ierr);
  // delete device data
  ierr = PetscLogEventBegin(events[5],0,0,0,0);CHKERRQ(ierr);
  CUDA_SAFE_CALL(hipFree(d_IPDataRaw));
  CUDA_SAFE_CALL(hipFree(d_invJj));
  CUDA_SAFE_CALL(hipFree(d_nu_alpha));
  CUDA_SAFE_CALL(hipFree(d_nu_beta));
  CUDA_SAFE_CALL(hipFree(d_invMass));
  CUDA_SAFE_CALL(hipFree(d_Eq_m));
  CUDA_SAFE_CALL(hipFree(d_BB));
  CUDA_SAFE_CALL(hipFree(d_DD));
  CUDA_SAFE_CALL(hipFree(d_f));
  CUDA_SAFE_CALL(hipFree(d_dfdx));
  CUDA_SAFE_CALL(hipFree(d_dfdy));
#if LANDAU_DIM==3
  CUDA_SAFE_CALL(hipFree(d_dfdz));
#endif
  //cerr = WaitForCUDA();CHKERRCUDA(cerr);
  ierr = PetscLogEventEnd(events[5],0,0,0,0);CHKERRQ(ierr);
  // First time assembly even with GPU assembly
  if (d_elemMats) {
    PetscScalar *elemMats=NULL,*elMat;
    ierr = PetscLogEventBegin(events[5],0,0,0,0);CHKERRQ(ierr);
    ierr = PetscMalloc1(totDim*totDim*numGCells,&elemMats);CHKERRQ(ierr);
    CUDA_SAFE_CALL(hipMemcpy(elemMats, d_elemMats, totDim*totDim*numGCells*sizeof(PetscScalar), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipFree(d_elemMats));
    ierr = PetscLogEventEnd(events[5],0,0,0,0);CHKERRQ(ierr);
    ierr = PetscLogEventBegin(events[6],0,0,0,0);CHKERRQ(ierr);
    for (ej = cStart, elMat = elemMats ; ej < cEnd; ++ej, elMat += totDim*totDim) {
      ierr = DMPlexMatSetClosure(plex, section, globalSection, JacP, ej, elMat, ADD_VALUES);CHKERRQ(ierr);
      if (ej==-1) {
        int d,f;
        PetscPrintf(PETSC_COMM_SELF,"GPU Element matrix\n");
        for (d = 0; d < totDim; ++d){
          for (f = 0; f < totDim; ++f) PetscPrintf(PETSC_COMM_SELF," %12.5e",  PetscRealPart(elMat[d*totDim + f]));
          PetscPrintf(PETSC_COMM_SELF,"\n");
        }
      }
    }
    ierr = PetscFree(elemMats);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(events[6],0,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}
